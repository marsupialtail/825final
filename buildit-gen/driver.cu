#include "hip/hip_runtime.h"
#include <cnpy.h>
#include <iostream>
#include "matrix.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <hipblas.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

//void __global__ mm(const float * __restrict__ BC, const sparse_matrix AB, const float * __restrict__ bias, float *AC, float *offsets)
#include "gencode.inc"


int main(int argc, char* argv[]) {
	// argv[1] = sparse_matrix.data
	// argv[2] = BC file name
	// argv[3] = bias file name
	// argv[4] = ref.npy
	sparse_matrix AB;
	load_matrix(AB, argv[1]);
	
	cnpy::NpyArray arr1 = cnpy::npy_load(argv[2]);	
	float * BC = arr1.data<float>();
	assert(arr1.word_size == sizeof(float));
	assert(arr1.shape.size() == 2 && arr1.shape[0] == B_dim && arr1.shape[1] == C_dim);
	
	cnpy::NpyArray arr4 = cnpy::npy_load(argv[3]);
        float * bias = arr4.data<float>();
        assert(arr4.word_size = sizeof(float));	
	assert(arr4.shape.size()==1 && arr4.shape[0] == 128);	

	cnpy::NpyArray arr2 = cnpy::npy_load(argv[4]);
        float * AC = arr2.data<float>();
	
	float *d_BC, *d_AC, *d_bias;
	int *d_offsets;	
	sparse_matrix AB_d;

	hipMalloc((void**)&d_BC, B_dim * C_dim * sizeof(float));
	hipMalloc((void**)&d_AC, A_dim * C_dim * sizeof(float));
	
	AB_d.nnz = AB.nnz;	
	hipMalloc((void**)&AB_d.row_val, AB_d.nnz * sizeof(int));
	hipMalloc((void**)&AB_d.values, AB_d.nnz * sizeof(float));
	hipMalloc((void**)&d_offsets, offsets_size * sizeof(int));	
	hipMalloc((void**)&d_bias, A_dim * sizeof(int));	
	
	hipMemcpy(d_BC, BC, B_dim * C_dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_bias, bias, A_dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(AB_d.row_val, AB.row_val, AB.nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(AB_d.values, AB.values, AB.nnz * sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(d_offsets, offsets, offsets_size * sizeof(int), hipMemcpyHostToDevice);
	
	float *result;
	result = new float[A_dim * C_dim];
	
	dim3 BS(A_blocks, C_blocks);
	dim3 TS(Gy_i + Gy_d, C_dim/C_blocks);

	for (int i = 0; i < 1000; i++)	
		mm<<<BS, TS>>>(d_BC, AB_d, d_bias, d_AC, d_offsets);

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipProfilerStart();
	hipEventRecord(start);

	for(int i = 0;i < 1000;i ++){
		mm<<<BS, TS>>>(d_BC, AB_d, d_bias, d_AC, d_offsets);
	}

	hipEventRecord(stop);
	hipProfilerStop();
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	std::cout << "kernel used " << time / 1000.0 << std::endl;
	
	hipMemcpy(result, d_AC, A_dim * C_dim * sizeof(float), hipMemcpyDeviceToHost);
	float error = 0 ;
	for (int i = 0; i < A_dim * C_dim; i++) 
		error += std::abs(result[i] - AC[i]);
	//std::cout << result[0] << " " << result[1] << " " << result[2] << std::endl;
	//std::cout << AC[0] << " " << AC[1] << " " << AC[2] << std::endl;
	std::cout << error << std::endl;
	if (error > 0.01)
		return -1;
	return 0;
}
