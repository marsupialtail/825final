#include "hip/hip_runtime.h"

#include <cnpy.h>

#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <hipblas.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>
// we are doing AC = AB * BC, reduce across the B dimension
// binding B to the x dimension, A to the y dimension and C to the z dimension

#define Tsy 1
#define Tsz (3136 / 98)
#define Gsy Tsz
#define Gy 1
#define Block_size (Gy * Gsy)
#define In_Format 'NCHW'
#define Out_Format 'NCHW'

namespace cg = cooperative_groups;

__global__ void mm(const float * __restrict__ BC, float * AC)
{
    register float ACC[64] = {0.0};
	register float RC = 0.0;
#if Gy > 1	
        __shared__ float result[64][Tsz];
	for(int i = threadIdx.x; i < 64 * Tsz; i += Block_size)
	{
		((float*)result)[i] = 0.0;
	}
	__syncthreads();
#endif
#if In_Format == 'NHWC'
	__shared__ float smem_cache[Tsz][32+1];
#endif
#if Out_Format == 'NHWC'
	__shared__ float smem_result[Tsz][64+1];
#endif

	int A_offset = blockIdx.x * (128 / 2);
	int C_offset = blockIdx.y * (3136 / 98);
	int groupId = threadIdx.x / (Gsy);
	int lane = threadIdx.x % (Gsy);


if(blockIdx.x == 0)
{



	if(groupId == 0)
	{


		    RC = BC[0 + C_offset + lane];
    
		ACC[1] += RC * -0.18584795f;   

		ACC[2] += RC * 0.07350932f;   

		ACC[4] += RC * 1.0029237f;   

		ACC[27] += RC * 0.16351075f;   

		ACC[28] += RC * 0.05196784f;   

		ACC[29] += RC * -0.11776184f;   

		ACC[45] += RC * 0.9202288f;   

		ACC[46] += RC * -0.10351631f;   

		ACC[52] += RC * -0.19278231f;   

		    RC = BC[3136 + C_offset + lane];
    
		ACC[8] += RC * 0.49481606f;   

		ACC[17] += RC * 0.102857f;   

		ACC[24] += RC * 0.34323132f;   

		ACC[28] += RC * 0.11447914f;   

		ACC[30] += RC * 1.0406339f;   

		ACC[38] += RC * 0.12667674f;   

		ACC[62] += RC * -0.105981514f;   

		    RC = BC[6272 + C_offset + lane];
    
		ACC[10] += RC * -0.21451224f;   

		ACC[21] += RC * 0.34443143f;   

		ACC[46] += RC * -0.11600338f;   

		    RC = BC[9408 + C_offset + lane];
    
		ACC[21] += RC * -0.54315495f;   

		ACC[22] += RC * -0.32288057f;   

		    RC = BC[12544 + C_offset + lane];
    
		ACC[2] += RC * 0.035814032f;   

		ACC[29] += RC * 0.15621199f;   

		ACC[32] += RC * -0.29438356f;   

		ACC[34] += RC * 0.19884212f;   

		    RC = BC[18816 + C_offset + lane];
    
		ACC[13] += RC * -0.4240484f;   

		ACC[35] += RC * -0.59533787f;   

		ACC[62] += RC * 0.14454196f;   

		    RC = BC[21952 + C_offset + lane];
    
		ACC[0] += RC * -0.22699068f;   

		ACC[6] += RC * 0.13832037f;   

		ACC[19] += RC * -0.27171835f;   

		ACC[29] += RC * -0.16838886f;   

		ACC[31] += RC * 0.18778998f;   

		ACC[41] += RC * 0.1515109f;   

		ACC[42] += RC * -0.645049f;   

		ACC[43] += RC * -0.66751015f;   

		ACC[47] += RC * -0.13399652f;   

		ACC[54] += RC * -0.14735857f;   

		ACC[60] += RC * -0.2250801f;   

		ACC[62] += RC * 0.10988758f;   

		ACC[63] += RC * 0.121927194f;   

		    RC = BC[25088 + C_offset + lane];
    
		ACC[32] += RC * 0.39211276f;   

		ACC[51] += RC * 0.64756685f;   

		    RC = BC[28224 + C_offset + lane];
    
		ACC[9] += RC * 0.23206377f;   

		ACC[11] += RC * 0.355418f;   

		ACC[38] += RC * 0.081785716f;   

		    RC = BC[31360 + C_offset + lane];
    
		ACC[0] += RC * 0.32356438f;   

		ACC[3] += RC * -0.6257694f;   

		ACC[18] += RC * -0.36066723f;   

		ACC[19] += RC * 0.19064696f;   

		ACC[29] += RC * -0.18113218f;   

		ACC[34] += RC * 0.21888024f;   

		ACC[42] += RC * 0.4985783f;   

		ACC[43] += RC * 0.33376634f;   

		ACC[44] += RC * 0.1442427f;   

		    RC = BC[34496 + C_offset + lane];
    
		ACC[1] += RC * -0.12532695f;   

		ACC[6] += RC * 0.15466522f;   

		ACC[8] += RC * -0.35663697f;   

		ACC[11] += RC * -0.8898302f;   

		ACC[20] += RC * -0.18002282f;   

		ACC[29] += RC * 0.37181646f;   

		ACC[41] += RC * 0.18639983f;   

		ACC[46] += RC * -0.037677366f;   

		    RC = BC[37632 + C_offset + lane];
    
		ACC[2] += RC * 0.04931372f;   

		ACC[29] += RC * 0.2322457f;   

		ACC[34] += RC * -0.40378276f;   

		ACC[40] += RC * 0.2730293f;   

		ACC[52] += RC * -0.19606051f;   

		    RC = BC[40768 + C_offset + lane];
    
		ACC[2] += RC * 0.033532735f;   

		ACC[3] += RC * -0.25232637f;   

		ACC[16] += RC * -0.33985403f;   

		ACC[18] += RC * -0.30187717f;   

		ACC[19] += RC * 0.24938029f;   

		ACC[46] += RC * 0.059925582f;   

		ACC[48] += RC * 0.19251324f;   

		ACC[54] += RC * -0.29366392f;   

		ACC[60] += RC * -0.34095109f;   

		    RC = BC[43904 + C_offset + lane];
    
		ACC[13] += RC * 0.3031487f;   

		ACC[25] += RC * -0.13325948f;   

		ACC[35] += RC * 0.7098635f;   

		ACC[44] += RC * 0.4846318f;   

		ACC[50] += RC * -0.8369425f;   

		ACC[57] += RC * 0.520863f;   

		    RC = BC[47040 + C_offset + lane];
    
		ACC[2] += RC * -0.043606438f;   

		ACC[4] += RC * 0.6640511f;   

		ACC[17] += RC * -0.12276016f;   

		ACC[20] += RC * -0.3290968f;   

		ACC[27] += RC * -0.13328187f;   

		ACC[29] += RC * -0.15515116f;   

		ACC[51] += RC * -0.34699997f;   

		ACC[52] += RC * 0.19347595f;   

		    RC = BC[50176 + C_offset + lane];
    
		ACC[31] += RC * -0.12047405f;   

		ACC[62] += RC * 0.22767809f;   

		ACC[63] += RC * -0.27400735f;   

		    RC = BC[53312 + C_offset + lane];
    
		ACC[9] += RC * -0.3624156f;   

		ACC[47] += RC * 0.104587376f;   

		ACC[48] += RC * -0.49369574f;   

		ACC[52] += RC * 0.31744704f;   

		ACC[56] += RC * -0.37888768f;   

		ACC[59] += RC * 0.27705458f;   

		ACC[61] += RC * -0.59206426f;   

		ACC[62] += RC * -0.16184911f;   

		ACC[63] += RC * -0.18657786f;   

		    RC = BC[56448 + C_offset + lane];
    
		ACC[14] += RC * 0.21536787f;   

		ACC[31] += RC * 0.49074826f;   

		ACC[53] += RC * -0.36852106f;   

		ACC[55] += RC * -0.53320783f;   

		ACC[58] += RC * -0.40168795f;   

		ACC[62] += RC * 0.22343624f;   

		    RC = BC[59584 + C_offset + lane];
    
		ACC[1] += RC * -0.15016143f;   

		ACC[2] += RC * 0.04522205f;   

		ACC[9] += RC * 0.10925903f;   

		ACC[10] += RC * -0.3644416f;   

		ACC[11] += RC * 0.2997704f;   

		ACC[15] += RC * 0.2818224f;   

		ACC[21] += RC * 1.2042669f;   

		ACC[46] += RC * -0.13913985f;   

		ACC[52] += RC * -0.19981512f;   

		    RC = BC[62720 + C_offset + lane];
    
		ACC[13] += RC * -0.42296624f;   

		ACC[44] += RC * 0.19960028f;   

		ACC[53] += RC * -0.39095968f;   

		ACC[62] += RC * -0.14741841f;   

		    RC = BC[65856 + C_offset + lane];
    
		ACC[2] += RC * 0.13015383f;   

		ACC[10] += RC * -0.27304852f;   

		ACC[15] += RC * 0.12982921f;   

		ACC[17] += RC * -0.09169744f;   

		ACC[22] += RC * -0.11175012f;   

		ACC[29] += RC * -0.17992558f;   

		ACC[38] += RC * -0.10121323f;   

		ACC[39] += RC * 0.05587235f;   

		ACC[41] += RC * 0.25168929f;   

		ACC[46] += RC * -0.1194533f;   

		    RC = BC[68992 + C_offset + lane];
    
		ACC[2] += RC * 0.037355147f;   

		ACC[29] += RC * 0.24902283f;   

		    RC = BC[72128 + C_offset + lane];
    
		ACC[15] += RC * -0.28222182f;   

		ACC[46] += RC * 0.03862857f;   

		    RC = BC[75264 + C_offset + lane];
    
		ACC[17] += RC * 0.13590872f;   

		ACC[23] += RC * 0.9363806f;   

		ACC[25] += RC * 0.09702684f;   

		ACC[28] += RC * 0.066035986f;   

		ACC[44] += RC * 0.18337922f;   

		    RC = BC[78400 + C_offset + lane];
    
		ACC[17] += RC * 0.08865731f;   

		ACC[49] += RC * -0.6772053f;   

		    RC = BC[81536 + C_offset + lane];
    
		ACC[1] += RC * 0.085653275f;   

		ACC[4] += RC * 0.44263017f;   

		ACC[20] += RC * -0.21628965f;   

		ACC[32] += RC * -0.38544872f;   

		ACC[34] += RC * 0.37335655f;   

		ACC[40] += RC * -0.3120128f;   

		    RC = BC[84672 + C_offset + lane];
    
		ACC[0] += RC * -0.20678516f;   

		ACC[3] += RC * 0.32530537f;   

		ACC[7] += RC * -0.64888483f;   

		ACC[16] += RC * 0.24505733f;   

		ACC[18] += RC * 0.5256636f;   

		ACC[19] += RC * -0.3272166f;   

		ACC[22] += RC * -0.13409023f;   

		ACC[26] += RC * -0.17373987f;   

		ACC[31] += RC * 0.2050952f;   

		ACC[47] += RC * 0.11039164f;   

		ACC[48] += RC * -0.2266204f;   

		ACC[54] += RC * -0.21862362f;   

		ACC[59] += RC * 0.12699634f;   

		    RC = BC[87808 + C_offset + lane];
    
		ACC[9] += RC * 0.15279768f;   

		ACC[11] += RC * 0.16959256f;   

		ACC[16] += RC * -0.42881632f;   

		ACC[59] += RC * 0.14944461f;   

		    RC = BC[90944 + C_offset + lane];
    
		ACC[3] += RC * 0.33154777f;   

		ACC[9] += RC * -0.17489237f;   

		ACC[34] += RC * -0.23416986f;   

		ACC[38] += RC * -0.11405346f;   

		ACC[41] += RC * -0.121796f;   

		ACC[47] += RC * 0.16470617f;   

		ACC[52] += RC * 0.21700798f;   

		ACC[54] += RC * 0.15729876f;   

		ACC[56] += RC * 0.23263544f;   

		ACC[59] += RC * -0.21623065f;   

		ACC[61] += RC * 0.60681957f;   

		ACC[62] += RC * -0.2807839f;   

		ACC[63] += RC * 0.13468657f;   

		    RC = BC[94080 + C_offset + lane];
    
		ACC[15] += RC * -0.28389496f;   

		ACC[22] += RC * -0.23826738f;   

		ACC[46] += RC * 0.08869904f;   

		    RC = BC[97216 + C_offset + lane];
    
		ACC[15] += RC * -0.101051666f;   

		ACC[21] += RC * -0.20413171f;   

		ACC[46] += RC * 0.0930694f;   

		    RC = BC[100352 + C_offset + lane];
    
		ACC[49] += RC * -0.46746957f;   

		    RC = BC[103488 + C_offset + lane];
    
		ACC[7] += RC * -0.23927209f;   

		ACC[14] += RC * 0.39993063f;   

		ACC[28] += RC * 0.06608304f;   

		ACC[31] += RC * 0.41690415f;   

		ACC[41] += RC * -0.16268021f;   

		ACC[47] += RC * -0.06665658f;   

		ACC[54] += RC * 0.3898219f;   

		ACC[58] += RC * -0.4811902f;   

		ACC[62] += RC * -0.20769393f;   

		    RC = BC[106624 + C_offset + lane];
    
		ACC[4] += RC * 0.23032154f;   

		ACC[9] += RC * 0.23267138f;   

		ACC[18] += RC * 0.17405392f;   

		ACC[19] += RC * -0.30409917f;   

		ACC[26] += RC * -0.3498056f;   

		ACC[33] += RC * -0.41797748f;   

		ACC[37] += RC * -0.14948401f;   

		ACC[40] += RC * -0.7528111f;   

		ACC[47] += RC * -0.12718853f;   

		ACC[59] += RC * 0.30002242f;   

		ACC[60] += RC * 0.23654762f;   

		    RC = BC[109760 + C_offset + lane];
    
		ACC[2] += RC * 0.029297013f;   

		ACC[26] += RC * 0.14118852f;   

		ACC[28] += RC * -0.09510913f;   

		ACC[33] += RC * 0.24073741f;   

		ACC[46] += RC * -0.07880714f;   

		    RC = BC[112896 + C_offset + lane];
    
		ACC[13] += RC * 0.47092745f;   

		ACC[25] += RC * 0.14386271f;   

		ACC[35] += RC * 0.6366757f;   

		ACC[44] += RC * -0.27373043f;   

		ACC[50] += RC * -0.7100229f;   

		ACC[57] += RC * 0.60796255f;   

		    RC = BC[116032 + C_offset + lane];
    
		ACC[8] += RC * 0.25282413f;   

		ACC[18] += RC * -0.106563374f;   

		ACC[28] += RC * -0.16593163f;   

		ACC[41] += RC * 0.30251354f;   

		ACC[47] += RC * 0.13658065f;   

		ACC[54] += RC * -0.69706804f;   

		    RC = BC[119168 + C_offset + lane];
    
		ACC[19] += RC * -0.1432965f;   

		ACC[31] += RC * 0.29604664f;   

		ACC[34] += RC * -0.25980312f;   

		ACC[38] += RC * 0.12999922f;   

		ACC[47] += RC * -0.0854889f;   

		ACC[49] += RC * -0.29304054f;   

		ACC[54] += RC * -0.27289852f;   

		ACC[56] += RC * 0.20387344f;   

		ACC[58] += RC * 0.23806095f;   

		ACC[60] += RC * -0.18487847f;   

		ACC[63] += RC * 0.38124996f;   

		    RC = BC[122304 + C_offset + lane];
    
		ACC[22] += RC * -0.32027736f;   

		    RC = BC[125440 + C_offset + lane];
    
		ACC[1] += RC * -0.1730987f;   

		ACC[2] += RC * 0.06058641f;   

		ACC[11] += RC * -0.47734895f;   

		ACC[20] += RC * -0.3253965f;   

		ACC[27] += RC * 0.23532234f;   

		ACC[29] += RC * 0.28981793f;   

		ACC[32] += RC * 0.27740094f;   

		ACC[38] += RC * 0.08028157f;   

		ACC[41] += RC * 0.2635885f;   

		ACC[43] += RC * -0.30340588f;   

		ACC[46] += RC * -0.06796846f;   

		ACC[51] += RC * -0.3080269f;   

		ACC[52] += RC * -0.3164784f;   

		ACC[54] += RC * -0.1577772f;   

		    RC = BC[128576 + C_offset + lane];
    
		ACC[1] += RC * 0.17775148f;   

		ACC[2] += RC * -0.052164525f;   

		ACC[17] += RC * -0.14576583f;   

		ACC[29] += RC * -0.30311817f;   

		ACC[38] += RC * -0.10872306f;   

		ACC[39] += RC * 0.08131772f;   

		ACC[41] += RC * -0.18844928f;   

		ACC[51] += RC * 0.16056864f;   

		    RC = BC[131712 + C_offset + lane];
    
		ACC[6] += RC * 0.13167883f;   

		ACC[17] += RC * 0.098335385f;   

		ACC[31] += RC * -0.37775412f;   

		ACC[38] += RC * 0.1992557f;   

		ACC[39] += RC * -0.029296694f;   

		ACC[41] += RC * 0.142547f;   

		ACC[47] += RC * -0.08436086f;   

		ACC[49] += RC * -0.31818718f;   

		ACC[54] += RC * -0.4348505f;   

		ACC[55] += RC * 0.5758083f;   

		ACC[58] += RC * -0.31610832f;   

		ACC[62] += RC * -0.28211477f;   

		ACC[63] += RC * -0.088721775f;   

		    RC = BC[134848 + C_offset + lane];
    
		ACC[1] += RC * 0.16146538f;   

		ACC[2] += RC * -0.0475772f;   

		ACC[6] += RC * 0.3263237f;   

		ACC[8] += RC * -0.6276081f;   

		ACC[9] += RC * 0.11742934f;   

		ACC[10] += RC * 0.20307045f;   

		ACC[17] += RC * 0.12657759f;   

		ACC[28] += RC * 0.10099993f;   

		ACC[29] += RC * -0.24853194f;   

		ACC[38] += RC * 0.13664165f;   

		ACC[39] += RC * -0.059467845f;   

		ACC[41] += RC * 0.27962637f;   

		ACC[47] += RC * -0.33069918f;   

		ACC[62] += RC * 0.4070311f;   

		    RC = BC[137984 + C_offset + lane];
    
		ACC[5] += RC * -0.5277919f;   

		ACC[13] += RC * 0.4151213f;   

		ACC[23] += RC * 0.3098378f;   

		ACC[50] += RC * 0.60081923f;   

		ACC[53] += RC * 0.9438657f;   

		ACC[57] += RC * -0.48226568f;   

		ACC[62] += RC * -0.12101145f;   

		    RC = BC[141120 + C_offset + lane];
    
		ACC[1] += RC * -0.22886378f;   

		ACC[2] += RC * 0.12361719f;   

		ACC[4] += RC * -0.72761476f;   

		ACC[20] += RC * 0.07841185f;   

		ACC[27] += RC * 0.1727555f;   

		ACC[29] += RC * 0.22075741f;   

		ACC[32] += RC * 0.24208428f;   

		ACC[38] += RC * 0.18292496f;   

		ACC[41] += RC * 0.30513752f;   

		ACC[45] += RC * -0.2635817f;   

		ACC[46] += RC * -0.0830069f;   

		ACC[52] += RC * -0.44706273f;   

		ACC[54] += RC * -0.17014933f;   

		    RC = BC[144256 + C_offset + lane];
    
		ACC[12] += RC * 0.2964925f;   

		ACC[29] += RC * 0.26264462f;   

		ACC[32] += RC * 0.14424336f;   

		    RC = BC[147392 + C_offset + lane];
    
		ACC[16] += RC * 0.27318072f;   

		ACC[18] += RC * -0.17249444f;   

		ACC[19] += RC * 0.1799758f;   

		ACC[42] += RC * 0.37337103f;   

		ACC[54] += RC * -0.13773881f;   

		ACC[60] += RC * -0.15917754f;   

		    RC = BC[150528 + C_offset + lane];
    
		ACC[24] += RC * 0.8332341f;   

		ACC[25] += RC * -0.12432293f;   

		ACC[30] += RC * -1.3509405f;   

		ACC[44] += RC * 0.37730205f;   

		    RC = BC[153664 + C_offset + lane];
    
		ACC[17] += RC * -0.080590144f;   

		ACC[28] += RC * -0.067727126f;   

		ACC[38] += RC * -0.11767121f;   

		ACC[47] += RC * -0.053688377f;   

		    RC = BC[156800 + C_offset + lane];
    
		ACC[6] += RC * 0.17148225f;   

		ACC[19] += RC * 0.15576942f;   

		ACC[34] += RC * 0.24993496f;   

		ACC[43] += RC * 0.28970215f;   

		ACC[47] += RC * -0.049092744f;   

		ACC[48] += RC * -0.2749422f;   

		ACC[49] += RC * -0.482139f;   

		ACC[54] += RC * -0.21819173f;   

		ACC[56] += RC * -0.2958262f;   

		ACC[60] += RC * 0.1939095f;   

		    RC = BC[159936 + C_offset + lane];
    
		ACC[11] += RC * -0.20927271f;   

		ACC[16] += RC * -0.47554716f;   

		    RC = BC[163072 + C_offset + lane];
    
		ACC[25] += RC * -0.04016468f;   

		ACC[26] += RC * 0.23068126f;   

		ACC[33] += RC * 0.4181578f;   

		ACC[36] += RC * -1.1136609f;   

		ACC[37] += RC * 0.097754344f;   

		ACC[60] += RC * -0.32576373f;   

		    RC = BC[166208 + C_offset + lane];
    
		ACC[13] += RC * -0.30980462f;   

		ACC[23] += RC * -0.4268186f;   

		ACC[50] += RC * -0.28890938f;   

		ACC[57] += RC * 0.23223265f;   

		    RC = BC[172480 + C_offset + lane];
    
		ACC[1] += RC * -0.16658776f;   

		ACC[2] += RC * 0.0996416f;   

		ACC[9] += RC * 0.19291757f;   

		ACC[11] += RC * 0.63438296f;   

		ACC[20] += RC * -0.1459365f;   

		ACC[27] += RC * 0.11957474f;   

		ACC[29] += RC * -0.24668978f;   

		ACC[46] += RC * -0.08030703f;   

		ACC[52] += RC * -0.3264767f;   

		ACC[60] += RC * -0.21778284f;   

		    RC = BC[175616 + C_offset + lane];
    
		ACC[17] += RC * 0.19204687f;   

		ACC[38] += RC * 0.1554395f;   

		ACC[39] += RC * -0.047485f;   

		ACC[41] += RC * 0.11880568f;   

		ACC[54] += RC * -0.23030247f;   

		    RC = BC[178752 + C_offset + lane];
    
		ACC[12] += RC * -0.66184276f;   

		ACC[32] += RC * -0.33932766f;   

		ACC[42] += RC * -0.2929955f;   

		ACC[51] += RC * 0.20616594f;   

		ACC[52] += RC * 0.1678827f;   

		    RC = BC[181888 + C_offset + lane];
    
		ACC[2] += RC * 0.043896176f;   

		ACC[22] += RC * -0.20326512f;   

		ACC[26] += RC * -0.21282434f;   

		ACC[28] += RC * -0.08408172f;   

		ACC[33] += RC * -0.3274323f;   

		ACC[54] += RC * -0.23184451f;   

		ACC[60] += RC * 0.25018582f;   

		    RC = BC[185024 + C_offset + lane];
    
		ACC[15] += RC * -0.23927557f;   

		ACC[22] += RC * -0.112599134f;   

		ACC[46] += RC * 0.10714278f;   

		    RC = BC[188160 + C_offset + lane];
    
		ACC[2] += RC * -0.06315396f;   

		ACC[9] += RC * 0.3384536f;   

		ACC[22] += RC * 0.15122199f;   

		ACC[29] += RC * -0.27005216f;   

		ACC[31] += RC * -0.11214516f;   

		ACC[41] += RC * 0.27858534f;   

		ACC[52] += RC * -0.15154484f;   

		ACC[54] += RC * -0.2562287f;   

		ACC[63] += RC * -0.10051341f;   

		    RC = BC[191296 + C_offset + lane];
    
		ACC[2] += RC * -0.07809353f;   

		ACC[6] += RC * 0.12629469f;   

		ACC[10] += RC * 0.25280967f;   

		ACC[15] += RC * -0.36020976f;   

		ACC[21] += RC * 0.6824156f;   

		ACC[29] += RC * -0.18479209f;   

		ACC[41] += RC * -0.10888942f;   

		ACC[46] += RC * 0.12745552f;   

		ACC[52] += RC * 0.16152337f;   

		ACC[54] += RC * 0.13135277f;   

		    RC = BC[194432 + C_offset + lane];
    
		ACC[2] += RC * 0.020466007f;   

		ACC[6] += RC * -0.27377614f;   

		ACC[7] += RC * 0.1725072f;   

		ACC[21] += RC * 0.7983611f;   

		ACC[28] += RC * -0.088834435f;   

		ACC[29] += RC * 0.3583734f;   

		ACC[38] += RC * -0.08311054f;   

		ACC[52] += RC * -0.3431529f;   

		ACC[54] += RC * -0.22426634f;   

		    RC = BC[197568 + C_offset + lane];
    
		ACC[1] += RC * 0.13151796f;   

		ACC[17] += RC * -0.07872371f;   

		ACC[28] += RC * -0.08544529f;   

		ACC[39] += RC * 0.08887849f;   

	}



        AC[0 + C_offset  + lane] = max(ACC[0] + 3.977744f,0.0f);

        AC[3136 + C_offset  + lane] = max(ACC[1] + 6.530452f,0.0f);

        AC[6272 + C_offset  + lane] = max(ACC[2] + 0.6891806f,0.0f);

        AC[9408 + C_offset  + lane] = max(ACC[3] + 1.9361705f,0.0f);

        AC[12544 + C_offset  + lane] = max(ACC[4] + -3.6765223f,0.0f);

        AC[15680 + C_offset  + lane] = max(ACC[5] + 5.3994093f,0.0f);

        AC[18816 + C_offset  + lane] = max(ACC[6] + 2.343494f,0.0f);

        AC[21952 + C_offset  + lane] = max(ACC[7] + 4.982128f,0.0f);

        AC[25088 + C_offset  + lane] = max(ACC[8] + -0.9497072f,0.0f);

        AC[28224 + C_offset  + lane] = max(ACC[9] + -0.8413665f,0.0f);

        AC[31360 + C_offset  + lane] = max(ACC[10] + 10.717709f,0.0f);

        AC[34496 + C_offset  + lane] = max(ACC[11] + -0.6981136f,0.0f);

        AC[37632 + C_offset  + lane] = max(ACC[12] + 2.9473062f,0.0f);

        AC[40768 + C_offset  + lane] = max(ACC[13] + -1.6245604f,0.0f);

        AC[43904 + C_offset  + lane] = max(ACC[14] + 0.061071157f,0.0f);

        AC[47040 + C_offset  + lane] = max(ACC[15] + -2.6851745f,0.0f);

        AC[50176 + C_offset  + lane] = max(ACC[16] + 2.9532514f,0.0f);

        AC[53312 + C_offset  + lane] = max(ACC[17] + 2.908723f,0.0f);

        AC[56448 + C_offset  + lane] = max(ACC[18] + 0.58572805f,0.0f);

        AC[59584 + C_offset  + lane] = max(ACC[19] + 2.4610727f,0.0f);

        AC[62720 + C_offset  + lane] = max(ACC[20] + 5.1531906f,0.0f);

        AC[65856 + C_offset  + lane] = max(ACC[21] + -21.59833f,0.0f);

        AC[68992 + C_offset  + lane] = max(ACC[22] + 6.769128f,0.0f);

        AC[72128 + C_offset  + lane] = max(ACC[23] + -1.4368893f,0.0f);

        AC[75264 + C_offset  + lane] = max(ACC[24] + -2.4415185f,0.0f);

        AC[78400 + C_offset  + lane] = max(ACC[25] + 2.3887634f,0.0f);

        AC[81536 + C_offset  + lane] = max(ACC[26] + 4.2395906f,0.0f);

        AC[84672 + C_offset  + lane] = max(ACC[27] + 1.7414098f,0.0f);

        AC[87808 + C_offset  + lane] = max(ACC[28] + 4.020364f,0.0f);

        AC[90944 + C_offset  + lane] = max(ACC[29] + 3.3077111f,0.0f);

        AC[94080 + C_offset  + lane] = max(ACC[30] + 1.8318611f,0.0f);

        AC[97216 + C_offset  + lane] = max(ACC[31] + -3.4719281f,0.0f);

        AC[100352 + C_offset  + lane] = max(ACC[32] + 1.4029231f,0.0f);

        AC[103488 + C_offset  + lane] = max(ACC[33] + 2.0068154f,0.0f);

        AC[106624 + C_offset  + lane] = max(ACC[34] + 0.9218654f,0.0f);

        AC[109760 + C_offset  + lane] = max(ACC[35] + -6.0270934f,0.0f);

        AC[112896 + C_offset  + lane] = max(ACC[36] + 3.3614318f,0.0f);

        AC[116032 + C_offset  + lane] = max(ACC[37] + 2.8095539f,0.0f);

        AC[119168 + C_offset  + lane] = max(ACC[38] + 3.403461f,0.0f);

        AC[122304 + C_offset  + lane] = max(ACC[39] + 8.1054325f,0.0f);

        AC[125440 + C_offset  + lane] = max(ACC[40] + 3.8145614f,0.0f);

        AC[128576 + C_offset  + lane] = max(ACC[41] + -7.17782f,0.0f);

        AC[131712 + C_offset  + lane] = max(ACC[42] + 1.0847245f,0.0f);

        AC[134848 + C_offset  + lane] = max(ACC[43] + 1.5190965f,0.0f);

        AC[137984 + C_offset  + lane] = max(ACC[44] + -1.1734383f,0.0f);

        AC[141120 + C_offset  + lane] = max(ACC[45] + 0.07577264f,0.0f);

        AC[144256 + C_offset  + lane] = max(ACC[46] + 7.681935f,0.0f);

        AC[147392 + C_offset  + lane] = max(ACC[47] + 4.0016623f,0.0f);

        AC[150528 + C_offset  + lane] = max(ACC[48] + 4.7618046f,0.0f);

        AC[153664 + C_offset  + lane] = max(ACC[49] + 8.247117f,0.0f);

        AC[156800 + C_offset  + lane] = max(ACC[50] + 10.89353f,0.0f);

        AC[159936 + C_offset  + lane] = max(ACC[51] + 1.5308125f,0.0f);

        AC[163072 + C_offset  + lane] = max(ACC[52] + 5.9652867f,0.0f);

        AC[166208 + C_offset  + lane] = max(ACC[53] + 2.222279f,0.0f);

        AC[169344 + C_offset  + lane] = max(ACC[54] + 16.071999f,0.0f);

        AC[172480 + C_offset  + lane] = max(ACC[55] + 4.40592f,0.0f);

        AC[175616 + C_offset  + lane] = max(ACC[56] + 3.53694f,0.0f);

        AC[178752 + C_offset  + lane] = max(ACC[57] + -6.28261f,0.0f);

        AC[181888 + C_offset  + lane] = max(ACC[58] + 4.730913f,0.0f);

        AC[185024 + C_offset  + lane] = max(ACC[59] + 0.80060506f,0.0f);

        AC[188160 + C_offset  + lane] = max(ACC[60] + 4.3881507f,0.0f);

        AC[191296 + C_offset  + lane] = max(ACC[61] + 2.3128614f,0.0f);

        AC[194432 + C_offset  + lane] = max(ACC[62] + 4.072897f,0.0f);

        AC[197568 + C_offset  + lane] = max(ACC[63] + 4.2837954f,0.0f);

}

if(blockIdx.x == 1)
{



	if(groupId == 0)
	{


		    RC = BC[0 + C_offset + lane];
    
		ACC[62] += RC * -0.4406303f;   

		    RC = BC[3136 + C_offset + lane];
    
		ACC[22] += RC * 0.091088176f;   

		ACC[36] += RC * 0.55848783f;   

		ACC[46] += RC * 0.20598467f;   

		ACC[50] += RC * 0.22028092f;   

		ACC[53] += RC * 0.12283251f;   

		ACC[54] += RC * 0.13666105f;   

		    RC = BC[6272 + C_offset + lane];
    
		ACC[5] += RC * 0.20815681f;   

		ACC[18] += RC * -0.16649343f;   

		ACC[19] += RC * -0.5775647f;   

		ACC[24] += RC * 0.18504068f;   

		ACC[27] += RC * -0.57794267f;   

		ACC[38] += RC * -0.18265578f;   

		ACC[42] += RC * -0.2359908f;   

		ACC[44] += RC * 0.26388422f;   

		ACC[53] += RC * -0.08718044f;   

		ACC[63] += RC * 0.20574354f;   

		    RC = BC[9408 + C_offset + lane];
    
		ACC[8] += RC * 1.0667694f;   

		ACC[16] += RC * -0.12162542f;   

		ACC[61] += RC * -0.24287847f;   

		ACC[63] += RC * 0.08633752f;   

		    RC = BC[12544 + C_offset + lane];
    
		ACC[3] += RC * -0.6231197f;   

		ACC[38] += RC * -0.41663548f;   

		ACC[48] += RC * 0.13058908f;   

		    RC = BC[18816 + C_offset + lane];
    
		ACC[15] += RC * -0.08632509f;   

		    RC = BC[21952 + C_offset + lane];
    
		ACC[1] += RC * 0.14340843f;   

		ACC[4] += RC * -0.30484268f;   

		ACC[6] += RC * 0.36825827f;   

		ACC[10] += RC * 0.1624523f;   

		ACC[18] += RC * -0.32090935f;   

		ACC[21] += RC * -0.061324574f;   

		ACC[23] += RC * -0.15628628f;   

		ACC[24] += RC * 0.21420263f;   

		ACC[34] += RC * 0.3754557f;   

		ACC[54] += RC * 0.07404963f;   

		    RC = BC[25088 + C_offset + lane];
    
		ACC[12] += RC * 0.40567666f;   

		ACC[17] += RC * 0.23251288f;   

		ACC[30] += RC * -0.3227378f;   

		ACC[36] += RC * 0.2640671f;   

		ACC[41] += RC * -0.896036f;   

		ACC[59] += RC * -0.12072415f;   

		ACC[60] += RC * 0.32684088f;   

		    RC = BC[28224 + C_offset + lane];
    
		ACC[5] += RC * -0.52059394f;   

		ACC[11] += RC * 0.18056181f;   

		ACC[17] += RC * -0.36361873f;   

		ACC[18] += RC * 0.26469594f;   

		ACC[19] += RC * 0.5603244f;   

		ACC[24] += RC * -0.48774025f;   

		ACC[38] += RC * 0.2895176f;   

		ACC[42] += RC * 0.16287552f;   

		ACC[44] += RC * -0.8786358f;   

		ACC[53] += RC * 0.052632283f;   

		    RC = BC[31360 + C_offset + lane];
    
		ACC[1] += RC * 0.25844523f;   

		ACC[12] += RC * 0.3678633f;   

		ACC[18] += RC * -0.20652992f;   

		ACC[21] += RC * 0.02860625f;   

		ACC[23] += RC * 0.20161776f;   

		ACC[49] += RC * 0.64013994f;   

		    RC = BC[34496 + C_offset + lane];
    
		ACC[5] += RC * 0.33973366f;   

		ACC[18] += RC * -0.28541043f;   

		ACC[19] += RC * -0.5211075f;   

		ACC[24] += RC * 0.28496334f;   

		ACC[38] += RC * 0.4118586f;   

		ACC[44] += RC * 0.34815535f;   

		ACC[53] += RC * 0.07106397f;   

		    RC = BC[37632 + C_offset + lane];
    
		ACC[3] += RC * -0.6908116f;   

		ACC[38] += RC * -0.4022358f;   

		ACC[48] += RC * 0.18182096f;   

		    RC = BC[40768 + C_offset + lane];
    
		ACC[2] += RC * -0.27051896f;   

		ACC[4] += RC * 0.26418802f;   

		ACC[6] += RC * -0.22417636f;   

		ACC[13] += RC * 0.11076752f;   

		ACC[16] += RC * -0.2352699f;   

		ACC[22] += RC * -0.06142882f;   

		ACC[25] += RC * -0.23517811f;   

		ACC[28] += RC * 0.33756998f;   

		ACC[37] += RC * 0.3553626f;   

		ACC[39] += RC * -0.3641682f;   

		ACC[48] += RC * 0.38881904f;   

		ACC[57] += RC * 0.066223145f;   

		    RC = BC[43904 + C_offset + lane];
    
		ACC[30] += RC * 0.4237843f;   

		ACC[33] += RC * 0.45383757f;   

		ACC[36] += RC * -0.36284006f;   

		    RC = BC[47040 + C_offset + lane];
    
		ACC[41] += RC * 0.84823585f;   

		ACC[44] += RC * -0.20835146f;   

		ACC[60] += RC * -0.2641248f;   

		    RC = BC[50176 + C_offset + lane];
    
		ACC[14] += RC * -0.3333359f;   

		ACC[21] += RC * -0.18576123f;   

		ACC[29] += RC * 0.27656603f;   

		ACC[31] += RC * -0.09049547f;   

		ACC[43] += RC * -0.601761f;   

		ACC[57] += RC * -0.14112867f;   

		    RC = BC[53312 + C_offset + lane];
    
		ACC[2] += RC * 0.40836215f;   

		ACC[11] += RC * -0.12327007f;   

		ACC[17] += RC * 0.7166563f;   

		ACC[21] += RC * 0.14835687f;   

		ACC[23] += RC * 0.24019587f;   

		ACC[24] += RC * 0.23985353f;   

		ACC[34] += RC * -0.44319856f;   

		ACC[36] += RC * 0.49440685f;   

		ACC[40] += RC * 0.19167854f;   

		ACC[41] += RC * 0.307752f;   

		ACC[49] += RC * 0.26803735f;   

		    RC = BC[56448 + C_offset + lane];
    
		ACC[7] += RC * 0.6488399f;   

		ACC[11] += RC * 0.1301568f;   

		ACC[28] += RC * 0.30882934f;   

		ACC[29] += RC * -0.27747446f;   

		ACC[31] += RC * -0.22677533f;   

		ACC[57] += RC * 0.2715751f;   

		    RC = BC[59584 + C_offset + lane];
    
		ACC[5] += RC * -0.22435893f;   

		ACC[6] += RC * 0.41122705f;   

		ACC[11] += RC * 0.3052933f;   

		ACC[16] += RC * -0.15971515f;   

		ACC[19] += RC * 0.5101216f;   

		ACC[22] += RC * -0.067366935f;   

		ACC[24] += RC * -0.25120124f;   

		ACC[38] += RC * 0.14035158f;   

		ACC[40] += RC * -0.43715397f;   

		ACC[42] += RC * -0.17980106f;   

		ACC[51] += RC * 0.24254951f;   

		ACC[53] += RC * -0.07539134f;   

		ACC[63] += RC * 0.27311078f;   

		    RC = BC[62720 + C_offset + lane];
    
		ACC[26] += RC * 1.0373975f;   

		    RC = BC[65856 + C_offset + lane];
    
		ACC[1] += RC * 0.20897873f;   

		ACC[5] += RC * 0.47787747f;   

		ACC[13] += RC * -0.14578928f;   

		ACC[16] += RC * -0.1745981f;   

		ACC[18] += RC * -0.31983212f;   

		ACC[19] += RC * -0.94006985f;   

		ACC[22] += RC * -0.07130075f;   

		ACC[24] += RC * 0.38313714f;   

		ACC[27] += RC * -0.67933214f;   

		ACC[38] += RC * -0.18523245f;   

		ACC[42] += RC * -0.32479095f;   

		ACC[44] += RC * 0.31064206f;   

		ACC[48] += RC * 0.19336684f;   

		ACC[51] += RC * 0.57325405f;   

		ACC[53] += RC * -0.07721908f;   

		ACC[63] += RC * 0.20828615f;   

		    RC = BC[68992 + C_offset + lane];
    
		ACC[3] += RC * -0.44929525f;   

		ACC[38] += RC * -0.4956f;   

		ACC[48] += RC * 0.14596836f;   

		ACC[60] += RC * 0.16259946f;   

		    RC = BC[72128 + C_offset + lane];
    
		ACC[38] += RC * -0.23179027f;   

		ACC[40] += RC * 0.42678714f;   

		    RC = BC[75264 + C_offset + lane];
    
		ACC[15] += RC * -0.19233654f;   

		ACC[22] += RC * 0.07907943f;   

		ACC[46] += RC * -0.32640454f;   

		ACC[53] += RC * 0.02808538f;   

		ACC[56] += RC * 0.9160236f;   

		ACC[61] += RC * 0.15105776f;   

		    RC = BC[78400 + C_offset + lane];
    
		ACC[9] += RC * 0.36965203f;   

		ACC[33] += RC * 0.37337646f;   

		ACC[47] += RC * -0.13820614f;   

		ACC[50] += RC * 0.17192228f;   

		ACC[59] += RC * 0.1240179f;   

		    RC = BC[81536 + C_offset + lane];
    
		ACC[59] += RC * 0.08171398f;   

		    RC = BC[84672 + C_offset + lane];
    
		ACC[2] += RC * 0.5051878f;   

		ACC[4] += RC * -0.37206277f;   

		ACC[13] += RC * 0.1164334f;   

		ACC[16] += RC * -0.24667653f;   

		ACC[17] += RC * 0.25501922f;   

		ACC[25] += RC * 0.37876078f;   

		ACC[37] += RC * -0.30279717f;   

		ACC[39] += RC * -0.21880925f;   

		ACC[48] += RC * 0.32877222f;   

		ACC[49] += RC * -0.1759692f;   

		ACC[51] += RC * 0.15132824f;   

		ACC[52] += RC * 0.18719979f;   

		ACC[57] += RC * -0.074623965f;   

		    RC = BC[87808 + C_offset + lane];
    
		ACC[13] += RC * -0.07814742f;   

		ACC[24] += RC * -0.15904282f;   

		ACC[35] += RC * -1.0172532f;   

		ACC[38] += RC * -0.11877334f;   

		ACC[44] += RC * -0.315153f;   

		ACC[55] += RC * 0.26997954f;   

		    RC = BC[90944 + C_offset + lane];
    
		ACC[10] += RC * -0.38531703f;   

		ACC[11] += RC * -0.18128674f;   

		ACC[12] += RC * -0.4721898f;   

		ACC[18] += RC * 0.2796878f;   

		ACC[21] += RC * -0.053722873f;   

		ACC[23] += RC * -0.24814184f;   

		ACC[36] += RC * -0.2760433f;   

		ACC[37] += RC * 0.33000097f;   

		ACC[40] += RC * 0.15179594f;   

		ACC[47] += RC * 0.078985415f;   

		ACC[54] += RC * -0.12895069f;   

		    RC = BC[94080 + C_offset + lane];
    
		ACC[38] += RC * -0.23604664f;   

		ACC[40] += RC * 0.44253945f;   

		    RC = BC[97216 + C_offset + lane];
    
		ACC[6] += RC * 0.20294246f;   

		ACC[32] += RC * -0.47254777f;   

		ACC[38] += RC * -0.16003929f;   

		ACC[40] += RC * 0.30337873f;   

		ACC[61] += RC * 0.1879968f;   

		    RC = BC[100352 + C_offset + lane];
    
		ACC[9] += RC * 0.7538701f;   

		ACC[47] += RC * 0.09530357f;   

		    RC = BC[103488 + C_offset + lane];
    
		ACC[7] += RC * 0.530809f;   

		ACC[11] += RC * -0.16450137f;   

		ACC[14] += RC * -0.15800846f;   

		ACC[22] += RC * 0.063439f;   

		ACC[28] += RC * 0.35694954f;   

		ACC[29] += RC * -0.38846496f;   

		ACC[31] += RC * -0.20685743f;   

		ACC[39] += RC * 0.21267773f;   

		ACC[48] += RC * -0.19268388f;   

		ACC[52] += RC * 0.3178293f;   

		ACC[53] += RC * 0.0495657f;   

		ACC[54] += RC * 0.076075f;   

		ACC[57] += RC * 0.28270325f;   

		    RC = BC[106624 + C_offset + lane];
    
		ACC[18] += RC * 0.14951068f;   

		ACC[23] += RC * 0.17971806f;   

		ACC[24] += RC * -0.28865936f;   

		ACC[25] += RC * 0.37774938f;   

		ACC[37] += RC * -0.45645836f;   

		ACC[44] += RC * -0.19800922f;   

		ACC[56] += RC * -0.2593176f;   

		ACC[61] += RC * -0.13615313f;   

		    RC = BC[109760 + C_offset + lane];
    
		ACC[6] += RC * -0.4499957f;   

		ACC[8] += RC * 1.1393102f;   

		ACC[42] += RC * -0.13429818f;   

		ACC[48] += RC * 0.21427943f;   

		ACC[51] += RC * 0.26797965f;   

		ACC[53] += RC * -0.043698937f;   

		ACC[63] += RC * 0.096868806f;   

		    RC = BC[112896 + C_offset + lane];
    
		ACC[15] += RC * -0.12092155f;   

		ACC[26] += RC * -0.367689f;   

		ACC[33] += RC * 0.4610455f;   

		    RC = BC[116032 + C_offset + lane];
    
		ACC[10] += RC * 0.48143354f;   

		ACC[11] += RC * 0.1927379f;   

		ACC[13] += RC * 0.11634239f;   

		ACC[22] += RC * -0.13136567f;   

		ACC[39] += RC * -0.45201075f;   

		ACC[46] += RC * -0.14425245f;   

		ACC[48] += RC * 0.2557311f;   

		ACC[51] += RC * 0.33814302f;   

		ACC[53] += RC * -0.08074084f;   

		ACC[54] += RC * -0.16093913f;   

		ACC[63] += RC * 0.10440518f;   

		    RC = BC[119168 + C_offset + lane];
    
		ACC[9] += RC * -0.3006763f;   

		ACC[10] += RC * 0.34636855f;   

		ACC[11] += RC * 0.20608762f;   

		ACC[14] += RC * 0.24260342f;   

		ACC[17] += RC * -0.2925317f;   

		ACC[23] += RC * -0.20316848f;   

		ACC[43] += RC * 0.49689543f;   

		ACC[47] += RC * -0.22097862f;   

		ACC[52] += RC * -0.36790925f;   

		ACC[54] += RC * 0.11069383f;   

		    RC = BC[122304 + C_offset + lane];
    
		ACC[15] += RC * -0.08760723f;   

		ACC[32] += RC * -0.98680353f;   

		ACC[55] += RC * -0.6717163f;   

		ACC[61] += RC * -0.12629263f;   

		    RC = BC[125440 + C_offset + lane];
    
		ACC[0] += RC * -0.049236752f;   

		ACC[5] += RC * 0.31478804f;   

		ACC[12] += RC * -0.3559452f;   

		ACC[18] += RC * -0.2483207f;   

		ACC[19] += RC * -0.4347266f;   

		ACC[24] += RC * 0.32147488f;   

		ACC[42] += RC * -0.18268614f;   

		ACC[44] += RC * 0.5779853f;   

		ACC[48] += RC * 0.14837566f;   

		ACC[50] += RC * 0.15541449f;   

		ACC[53] += RC * 0.054397684f;   

		    RC = BC[128576 + C_offset + lane];
    
		ACC[1] += RC * -0.22827059f;   

		ACC[18] += RC * 0.16604872f;   

		ACC[22] += RC * -0.06136018f;   

		ACC[38] += RC * -0.21517621f;   

		ACC[50] += RC * -0.23421668f;   

		ACC[54] += RC * -0.09610469f;   

		    RC = BC[131712 + C_offset + lane];
    
		ACC[9] += RC * -0.45111597f;   

		ACC[10] += RC * 0.32330972f;   

		ACC[11] += RC * 0.25917482f;   

		ACC[14] += RC * -0.2415725f;   

		ACC[29] += RC * 0.28811523f;   

		ACC[47] += RC * -0.28767684f;   

		ACC[48] += RC * 0.1372894f;   

		ACC[50] += RC * 0.25730097f;   

		ACC[54] += RC * 0.1553135f;   

		ACC[59] += RC * 0.12625325f;   

		    RC = BC[134848 + C_offset + lane];
    
		ACC[6] += RC * -0.26206163f;   

		ACC[13] += RC * -0.13643606f;   

		ACC[14] += RC * -0.16992103f;   

		ACC[16] += RC * 0.23800203f;   

		ACC[21] += RC * -0.09796318f;   

		ACC[22] += RC * 0.055083614f;   

		ACC[42] += RC * 0.14952502f;   

		ACC[47] += RC * -0.29935917f;   

		ACC[48] += RC * -0.18766308f;   

		ACC[51] += RC * -0.31803492f;   

		ACC[61] += RC * -0.0584118f;   

		ACC[62] += RC * -0.6801283f;   

		    RC = BC[137984 + C_offset + lane];
    
		ACC[26] += RC * -0.38279307f;   

		ACC[33] += RC * 0.4642472f;   

		ACC[43] += RC * 0.48076943f;   

		ACC[45] += RC * 0.8154967f;   

		ACC[46] += RC * -0.49401584f;   

		ACC[56] += RC * -0.33043376f;   

		    RC = BC[141120 + C_offset + lane];
    
		ACC[0] += RC * -0.08347109f;   

		ACC[10] += RC * 0.27885252f;   

		ACC[11] += RC * 0.25426716f;   

		ACC[48] += RC * 0.26862422f;   

		ACC[53] += RC * 0.069793165f;   

		ACC[54] += RC * 0.112700365f;   

		    RC = BC[144256 + C_offset + lane];
    
		ACC[38] += RC * -0.39415953f;   

		ACC[59] += RC * -0.08427734f;   

		    RC = BC[147392 + C_offset + lane];
    
		ACC[1] += RC * -0.13147347f;   

		ACC[2] += RC * -0.23494175f;   

		ACC[4] += RC * 0.250107f;   

		ACC[13] += RC * 0.09344289f;   

		ACC[15] += RC * -0.19824103f;   

		ACC[28] += RC * 0.21619697f;   

		ACC[30] += RC * -0.5704185f;   

		ACC[31] += RC * -0.104502946f;   

		ACC[57] += RC * 0.051604852f;   

		ACC[61] += RC * 0.22274266f;   

		    RC = BC[150528 + C_offset + lane];
    
		ACC[30] += RC * 0.45899162f;   

		ACC[36] += RC * -0.5797182f;   

		    RC = BC[153664 + C_offset + lane];
    
		ACC[1] += RC * 0.15612988f;   

		ACC[5] += RC * -0.29628477f;   

		ACC[18] += RC * 0.24801366f;   

		ACC[22] += RC * -0.06886547f;   

		ACC[24] += RC * -0.15472333f;   

		ACC[38] += RC * -0.28598753f;   

		    RC = BC[156800 + C_offset + lane];
    
		ACC[9] += RC * -0.41365308f;   

		ACC[11] += RC * 0.16044457f;   

		ACC[17] += RC * 0.28503844f;   

		ACC[21] += RC * 0.1494196f;   

		ACC[23] += RC * 0.32455277f;   

		ACC[31] += RC * 0.113498814f;   

		ACC[47] += RC * -0.17516118f;   

		ACC[57] += RC * 0.12623154f;   

		    RC = BC[159936 + C_offset + lane];
    
		ACC[1] += RC * 0.65042615f;   

		ACC[6] += RC * 0.42594796f;   

		ACC[18] += RC * -0.13597205f;   

		    RC = BC[163072 + C_offset + lane];
    
		ACC[2] += RC * -0.42983758f;   

		ACC[12] += RC * -0.21008573f;   

		ACC[21] += RC * -0.054850865f;   

		ACC[24] += RC * -0.18604255f;   

		ACC[25] += RC * -0.30103827f;   

		ACC[37] += RC * 0.34806022f;   

		ACC[44] += RC * -0.15596001f;   

		ACC[52] += RC * -0.21316518f;   

		ACC[55] += RC * 0.37439975f;   

		    RC = BC[166208 + C_offset + lane];
    
		ACC[46] += RC * 0.42773876f;   

		    RC = BC[172480 + C_offset + lane];
    
		ACC[0] += RC * -0.07687451f;   

		ACC[3] += RC * -0.53101f;   

		ACC[5] += RC * -0.44122908f;   

		ACC[11] += RC * 0.21450573f;   

		ACC[24] += RC * -0.24097344f;   

		ACC[27] += RC * -0.17830954f;   

		ACC[44] += RC * -0.6377197f;   

		ACC[53] += RC * 0.055405397f;   

		    RC = BC[175616 + C_offset + lane];
    
		ACC[9] += RC * -0.22386803f;   

		ACC[14] += RC * 0.24518697f;   

		ACC[21] += RC * 0.06983554f;   

		ACC[48] += RC * 0.19773705f;   

		ACC[50] += RC * 0.5686969f;   

		ACC[54] += RC * 0.117811844f;   

		ACC[59] += RC * 0.25699562f;   

		    RC = BC[178752 + C_offset + lane];
    
		ACC[34] += RC * 0.40574792f;   

		ACC[38] += RC * -0.21245319f;   

		ACC[41] += RC * -0.581772f;   

		ACC[59] += RC * 0.08565152f;   

		    RC = BC[181888 + C_offset + lane];
    
		ACC[8] += RC * 0.7309588f;   

		ACC[22] += RC * -0.09992223f;   

		ACC[24] += RC * 0.13385744f;   

		ACC[28] += RC * -0.36533684f;   

		ACC[39] += RC * -0.25117013f;   

		ACC[48] += RC * 0.25177416f;   

		ACC[49] += RC * -0.28545076f;   

		ACC[53] += RC * -0.06506008f;   

		ACC[61] += RC * -0.28616887f;   

		    RC = BC[185024 + C_offset + lane];
    
		ACC[38] += RC * -0.21065329f;   

		ACC[40] += RC * 0.49672085f;   

		    RC = BC[188160 + C_offset + lane];
    
		ACC[10] += RC * 0.48832548f;   

		ACC[13] += RC * 0.102604404f;   

		ACC[14] += RC * -0.2498683f;   

		ACC[16] += RC * -0.26095983f;   

		ACC[24] += RC * 0.17783828f;   

		ACC[39] += RC * -0.26379153f;   

		ACC[47] += RC * -0.11098959f;   

		ACC[48] += RC * 0.36261433f;   

		ACC[51] += RC * 0.7271661f;   

		    RC = BC[191296 + C_offset + lane];
    
		ACC[0] += RC * -0.0407001f;   

		ACC[5] += RC * 0.18842979f;   

		ACC[11] += RC * -0.19668639f;   

		ACC[13] += RC * 0.22303726f;   

		ACC[19] += RC * -0.48006082f;   

		ACC[24] += RC * 0.14501482f;   

		ACC[27] += RC * -0.39214194f;   

		ACC[40] += RC * -0.34364948f;   

		ACC[53] += RC * 0.07577099f;   

		ACC[63] += RC * -0.27511463f;   

		    RC = BC[194432 + C_offset + lane];
    
		ACC[0] += RC * 0.08058559f;   

		ACC[5] += RC * -0.51369154f;   

		ACC[11] += RC * 0.29554477f;   

		ACC[16] += RC * -0.14971066f;   

		ACC[17] += RC * -0.2622464f;   

		ACC[18] += RC * 0.31855294f;   

		ACC[19] += RC * 1.2430506f;   

		ACC[22] += RC * -0.06381879f;   

		ACC[24] += RC * -0.4359843f;   

		ACC[27] += RC * 0.56794626f;   

		ACC[38] += RC * -0.17689636f;   

		ACC[44] += RC * -0.3349003f;   

		ACC[48] += RC * 0.3092569f;   

		ACC[53] += RC * -0.07257747f;   

		ACC[63] += RC * 0.22388396f;   

		    RC = BC[197568 + C_offset + lane];
    
		ACC[5] += RC * 0.17898779f;   

		ACC[18] += RC * -0.19749318f;   

		ACC[22] += RC * -0.051323816f;   

		ACC[24] += RC * 0.22011934f;   

		ACC[38] += RC * -0.3539517f;   

		ACC[54] += RC * -0.10591654f;   

	}



        AC[200704 + C_offset  + lane] = max(ACC[0] + 3.162181f,0.0f);

        AC[203840 + C_offset  + lane] = max(ACC[1] + -2.5823455f,0.0f);

        AC[206976 + C_offset  + lane] = max(ACC[2] + -2.7919135f,0.0f);

        AC[210112 + C_offset  + lane] = max(ACC[3] + 1.7371584f,0.0f);

        AC[213248 + C_offset  + lane] = max(ACC[4] + 3.1848283f,0.0f);

        AC[216384 + C_offset  + lane] = max(ACC[5] + -0.24944139f,0.0f);

        AC[219520 + C_offset  + lane] = max(ACC[6] + 0.014434099f,0.0f);

        AC[222656 + C_offset  + lane] = max(ACC[7] + -2.940038f,0.0f);

        AC[225792 + C_offset  + lane] = max(ACC[8] + -9.4151325f,0.0f);

        AC[228928 + C_offset  + lane] = max(ACC[9] + -3.1973352f,0.0f);

        AC[232064 + C_offset  + lane] = max(ACC[10] + -5.6700807f,0.0f);

        AC[235200 + C_offset  + lane] = max(ACC[11] + -6.306308f,0.0f);

        AC[238336 + C_offset  + lane] = max(ACC[12] + 1.8398961f,0.0f);

        AC[241472 + C_offset  + lane] = max(ACC[13] + 1.7022657f,0.0f);

        AC[244608 + C_offset  + lane] = max(ACC[14] + 5.3732634f,0.0f);

        AC[247744 + C_offset  + lane] = max(ACC[15] + 4.751031f,0.0f);

        AC[250880 + C_offset  + lane] = max(ACC[16] + 11.078623f,0.0f);

        AC[254016 + C_offset  + lane] = max(ACC[17] + -2.0852878f,0.0f);

        AC[257152 + C_offset  + lane] = max(ACC[18] + 8.101216f,0.0f);

        AC[260288 + C_offset  + lane] = max(ACC[19] + 6.377058f,0.0f);

        AC[263424 + C_offset  + lane] = max(ACC[20] + 2.3062682f,0.0f);

        AC[266560 + C_offset  + lane] = max(ACC[21] + 2.4351485f,0.0f);

        AC[269696 + C_offset  + lane] = max(ACC[22] + 5.1611986f,0.0f);

        AC[272832 + C_offset  + lane] = max(ACC[23] + 3.1691751f,0.0f);

        AC[275968 + C_offset  + lane] = max(ACC[24] + 0.50136256f,0.0f);

        AC[279104 + C_offset  + lane] = max(ACC[25] + -0.3714794f,0.0f);

        AC[282240 + C_offset  + lane] = max(ACC[26] + 3.2185884f,0.0f);

        AC[285376 + C_offset  + lane] = max(ACC[27] + 15.205744f,0.0f);

        AC[288512 + C_offset  + lane] = max(ACC[28] + -2.0926886f,0.0f);

        AC[291648 + C_offset  + lane] = max(ACC[29] + 3.2758942f,0.0f);

        AC[294784 + C_offset  + lane] = max(ACC[30] + -1.7504005f,0.0f);

        AC[297920 + C_offset  + lane] = max(ACC[31] + 5.1391497f,0.0f);

        AC[301056 + C_offset  + lane] = max(ACC[32] + 3.60483f,0.0f);

        AC[304192 + C_offset  + lane] = max(ACC[33] + -5.7226353f,0.0f);

        AC[307328 + C_offset  + lane] = max(ACC[34] + 2.104572f,0.0f);

        AC[310464 + C_offset  + lane] = max(ACC[35] + 2.8668141f,0.0f);

        AC[313600 + C_offset  + lane] = max(ACC[36] + -0.16970116f,0.0f);

        AC[316736 + C_offset  + lane] = max(ACC[37] + -0.23280361f,0.0f);

        AC[319872 + C_offset  + lane] = max(ACC[38] + 11.146543f,0.0f);

        AC[323008 + C_offset  + lane] = max(ACC[39] + 9.825462f,0.0f);

        AC[326144 + C_offset  + lane] = max(ACC[40] + 2.9263065f,0.0f);

        AC[329280 + C_offset  + lane] = max(ACC[41] + 2.70706f,0.0f);

        AC[332416 + C_offset  + lane] = max(ACC[42] + 10.834305f,0.0f);

        AC[335552 + C_offset  + lane] = max(ACC[43] + 2.3022428f,0.0f);

        AC[338688 + C_offset  + lane] = max(ACC[44] + -1.3121259f,0.0f);

        AC[341824 + C_offset  + lane] = max(ACC[45] + 1.028136f,0.0f);

        AC[344960 + C_offset  + lane] = max(ACC[46] + 4.571438f,0.0f);

        AC[348096 + C_offset  + lane] = max(ACC[47] + 6.4234734f,0.0f);

        AC[351232 + C_offset  + lane] = max(ACC[48] + -14.035306f,0.0f);

        AC[354368 + C_offset  + lane] = max(ACC[49] + 1.1635358f,0.0f);

        AC[357504 + C_offset  + lane] = max(ACC[50] + 0.11034632f,0.0f);

        AC[360640 + C_offset  + lane] = max(ACC[51] + -16.26934f,0.0f);

        AC[363776 + C_offset  + lane] = max(ACC[52] + 1.2764599f,0.0f);

        AC[366912 + C_offset  + lane] = max(ACC[53] + 5.10857f,0.0f);

        AC[370048 + C_offset  + lane] = max(ACC[54] + 2.3454251f,0.0f);

        AC[373184 + C_offset  + lane] = max(ACC[55] + 1.9230678f,0.0f);

        AC[376320 + C_offset  + lane] = max(ACC[56] + 0.96652675f,0.0f);

        AC[379456 + C_offset  + lane] = max(ACC[57] + 1.0347203f,0.0f);

        AC[382592 + C_offset  + lane] = max(ACC[58] + 1.883526f,0.0f);

        AC[385728 + C_offset  + lane] = max(ACC[59] + 0.81246436f,0.0f);

        AC[388864 + C_offset  + lane] = max(ACC[60] + 1.2826749f,0.0f);

        AC[392000 + C_offset  + lane] = max(ACC[61] + 3.50684f,0.0f);

        AC[395136 + C_offset  + lane] = max(ACC[62] + 4.6486897f,0.0f);

        AC[398272 + C_offset  + lane] = max(ACC[63] + -5.8354063f,0.0f);

}

 
}
int main()
{

	std::cout << "Group size " << Gsy << std::endl;

	cnpy::NpyArray arr = cnpy::npy_load("mobilenet/contraction_1x1_1_transposed.npy");
	float * AB = arr.data<float>();
	assert(arr.word_size = sizeof(float));
	assert(arr.shape.size()==2 && arr.shape[0] == 64 && arr.shape[1] == 128); //transposed

	cnpy::NpyArray arr1 = cnpy::npy_load("BC.npy");
	float * BC = arr1.data<float>();
	assert(arr1.word_size = sizeof(float));
#if In_Format == 'NHWC'
	assert(arr1.shape.size()==2 && arr1.shape[0] == 3136 && arr1.shape[1] == 64);
#else
	assert(arr1.shape.size()==2 && arr1.shape[0] == 64 && arr1.shape[1] == 3136);
#endif
    cnpy::NpyArray arr2 = cnpy::npy_load("ref.npy");
	float * AC = arr2.data<float>();
    std::cout << AC[0] << std::endl;

	float *d_BC, *d_AC;
	hipMalloc((void**)&d_BC, 64 * 3136 *sizeof(float));
	hipMalloc((void**)&d_AC, 128 * 3136 *sizeof(float));


	hipMemcpy( d_BC,BC, 64 * 3136 *sizeof(float), hipMemcpyHostToDevice);

	float *result;
	result = (float *)malloc(128 * 3136 *sizeof(result));

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	dim3 GS(2,98);

     std::cout << "warning: sometimes you might want to fix the launch dimensions to 32" << std::endl;

    for(int i = 0;i < 1000;i ++){
	    mm<<<GS,Gsy>>>(d_BC,d_AC);
    }

	hipProfilerStart();
	hipEventRecord(start);

	for(int i = 0;i < 1000;i ++){
	    mm<<<GS,Gsy>>>(d_BC,d_AC);
    }
	hipEventRecord(stop);
	hipProfilerStop();
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	std::cout << "kernel used " << time / 1000.0 << std::endl;

	hipMemcpy(result, d_AC, 128 * 3136 *sizeof(float), hipMemcpyDeviceToHost);

	float error = 0;
	for(int i = 0 ; i < 128 * 3136; i ++)
	{
        error += abs(result[i] - AC[i]);
	}
	
	#if Out_Format == 'NCHW'
        cnpy::npy_save("result.npy",&result[0],{128,3136},"w");
    #else
        cnpy::npy_save("result.npy",&result[0],{3136,128},"w");
    #endif

	std::cout << result[0] << result[1] << result[2] << std::endl;
	std::cout << error << std::endl;
	hipFree(d_BC);
	hipFree(d_AC);
}
