#include "hip/hip_runtime.h"

#include <cnpy.h>

#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <hipblas.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>
// we are doing AC = AB * BC, reduce across the B dimension
// binding B to the x dimension, A to the y dimension and C to the z dimension

#define Tsy 1
#define Tsz (3136 / 98)
#define Gsy Tsz
#define Gy 1
#define Block_size (Gy * Gsy)
#define In_Format 'NCHW'
#define Out_Format 'NCHW'

namespace cg = cooperative_groups;

__global__ void mm(const float * __restrict__ BC, const float * __restrict__ BA, const float * __restrict__ bias, float * AC)
{
    register float ACC[128] = {0.0};
	register float RC = 0.0;
#if Gy > 1	
        __shared__ float result[128][Tsz];
	for(int i = threadIdx.x; i < 128 * Tsz; i += Block_size)
	{
		((float*)result)[i] = 0.0;
	}
	__syncthreads();
#endif
#if In_Format == 'NHWC'
	__shared__ float smem_cache[Tsz][32+1];
#endif
#if Out_Format == 'NHWC'
	__shared__ float smem_result[Tsz][128+1];
#endif

	int A_offset = blockIdx.x * (128 / 1);
	int C_offset = blockIdx.y * (3136 / 98);
	int groupId = threadIdx.x / (Gsy);
	int lane = threadIdx.x % (Gsy);


if(blockIdx.x == 0)
{


int A_offset = 0;
int block_NY = 128;

	if(groupId == 0)
	{


		    RC = BC[0 + C_offset + lane];
    
		ACC[1] += RC * -0.18584795f;

		ACC[2] += RC * 0.07350932f;

		ACC[4] += RC * 1.0029237f;

		ACC[27] += RC * 0.16351075f;

		ACC[28] += RC * 0.05196784f;

		ACC[29] += RC * -0.11776184f;

		ACC[45] += RC * 0.9202288f;

		ACC[46] += RC * -0.10351631f;

		ACC[52] += RC * -0.19278231f;

		    RC = BC[3136 + C_offset + lane];
    
		ACC[8] += RC * 0.49481606f;

		ACC[17] += RC * 0.102857f;

		ACC[24] += RC * 0.34323132f;

		ACC[28] += RC * 0.11447914f;

		ACC[30] += RC * 1.0406339f;

		ACC[38] += RC * 0.12667674f;

		ACC[62] += RC * -0.105981514f;

		    RC = BC[6272 + C_offset + lane];
    
		ACC[10] += RC * -0.21451224f;

		ACC[21] += RC * 0.34443143f;

		ACC[46] += RC * -0.11600338f;

		    RC = BC[9408 + C_offset + lane];
    
		ACC[21] += RC * -0.54315495f;

		ACC[22] += RC * -0.32288057f;

		    RC = BC[12544 + C_offset + lane];
    
		ACC[2] += RC * 0.035814032f;

		ACC[29] += RC * 0.15621199f;

		ACC[32] += RC * -0.29438356f;

		ACC[34] += RC * 0.19884212f;

		    RC = BC[18816 + C_offset + lane];
    
		ACC[13] += RC * -0.4240484f;

		ACC[35] += RC * -0.59533787f;

		ACC[62] += RC * 0.14454196f;

		    RC = BC[21952 + C_offset + lane];
    
		ACC[0] += RC * -0.22699068f;

		ACC[6] += RC * 0.13832037f;

		ACC[19] += RC * -0.27171835f;

		ACC[29] += RC * -0.16838886f;

		ACC[31] += RC * 0.18778998f;

		ACC[41] += RC * 0.1515109f;

		ACC[42] += RC * -0.645049f;

		ACC[43] += RC * -0.66751015f;

		ACC[47] += RC * -0.13399652f;

		ACC[54] += RC * -0.14735857f;

		ACC[60] += RC * -0.2250801f;

		ACC[62] += RC * 0.10988758f;

		ACC[63] += RC * 0.121927194f;

		    RC = BC[25088 + C_offset + lane];
    
		ACC[32] += RC * 0.39211276f;

		ACC[51] += RC * 0.64756685f;

		    RC = BC[28224 + C_offset + lane];
    
		ACC[9] += RC * 0.23206377f;

		ACC[11] += RC * 0.355418f;

		ACC[38] += RC * 0.081785716f;

		    RC = BC[31360 + C_offset + lane];
    
		ACC[0] += RC * 0.32356438f;

		ACC[3] += RC * -0.6257694f;

		ACC[18] += RC * -0.36066723f;

		ACC[19] += RC * 0.19064696f;

		ACC[29] += RC * -0.18113218f;

		ACC[34] += RC * 0.21888024f;

		ACC[42] += RC * 0.4985783f;

		ACC[43] += RC * 0.33376634f;

		ACC[44] += RC * 0.1442427f;

		    RC = BC[34496 + C_offset + lane];
    
		ACC[1] += RC * -0.12532695f;

		ACC[6] += RC * 0.15466522f;

		ACC[8] += RC * -0.35663697f;

		ACC[11] += RC * -0.8898302f;

		ACC[20] += RC * -0.18002282f;

		ACC[29] += RC * 0.37181646f;

		ACC[41] += RC * 0.18639983f;

		ACC[46] += RC * -0.037677366f;

		    RC = BC[37632 + C_offset + lane];
    
		ACC[2] += RC * 0.04931372f;

		ACC[29] += RC * 0.2322457f;

		ACC[34] += RC * -0.40378276f;

		ACC[40] += RC * 0.2730293f;

		ACC[52] += RC * -0.19606051f;

		    RC = BC[40768 + C_offset + lane];
    
		ACC[2] += RC * 0.033532735f;

		ACC[3] += RC * -0.25232637f;

		ACC[16] += RC * -0.33985403f;

		ACC[18] += RC * -0.30187717f;

		ACC[19] += RC * 0.24938029f;

		ACC[46] += RC * 0.059925582f;

		ACC[48] += RC * 0.19251324f;

		ACC[54] += RC * -0.29366392f;

		ACC[60] += RC * -0.34095109f;

		    RC = BC[43904 + C_offset + lane];
    
		ACC[13] += RC * 0.3031487f;

		ACC[25] += RC * -0.13325948f;

		ACC[35] += RC * 0.7098635f;

		ACC[44] += RC * 0.4846318f;

		ACC[50] += RC * -0.8369425f;

		ACC[57] += RC * 0.520863f;

		    RC = BC[47040 + C_offset + lane];
    
		ACC[2] += RC * -0.043606438f;

		ACC[4] += RC * 0.6640511f;

		ACC[17] += RC * -0.12276016f;

		ACC[20] += RC * -0.3290968f;

		ACC[27] += RC * -0.13328187f;

		ACC[29] += RC * -0.15515116f;

		ACC[51] += RC * -0.34699997f;

		ACC[52] += RC * 0.19347595f;

		    RC = BC[50176 + C_offset + lane];
    
		ACC[31] += RC * -0.12047405f;

		ACC[62] += RC * 0.22767809f;

		ACC[63] += RC * -0.27400735f;

		    RC = BC[53312 + C_offset + lane];
    
		ACC[9] += RC * -0.3624156f;

		ACC[47] += RC * 0.104587376f;

		ACC[48] += RC * -0.49369574f;

		ACC[52] += RC * 0.31744704f;

		ACC[56] += RC * -0.37888768f;

		ACC[59] += RC * 0.27705458f;

		ACC[61] += RC * -0.59206426f;

		ACC[62] += RC * -0.16184911f;

		ACC[63] += RC * -0.18657786f;

		    RC = BC[56448 + C_offset + lane];
    
		ACC[14] += RC * 0.21536787f;

		ACC[31] += RC * 0.49074826f;

		ACC[53] += RC * -0.36852106f;

		ACC[55] += RC * -0.53320783f;

		ACC[58] += RC * -0.40168795f;

		ACC[62] += RC * 0.22343624f;

		    RC = BC[59584 + C_offset + lane];
    
		ACC[1] += RC * -0.15016143f;

		ACC[2] += RC * 0.04522205f;

		ACC[9] += RC * 0.10925903f;

		ACC[10] += RC * -0.3644416f;

		ACC[11] += RC * 0.2997704f;

		ACC[15] += RC * 0.2818224f;

		ACC[21] += RC * 1.2042669f;

		ACC[46] += RC * -0.13913985f;

		ACC[52] += RC * -0.19981512f;

		    RC = BC[62720 + C_offset + lane];
    
		ACC[13] += RC * -0.42296624f;

		ACC[44] += RC * 0.19960028f;

		ACC[53] += RC * -0.39095968f;

		ACC[62] += RC * -0.14741841f;

		    RC = BC[65856 + C_offset + lane];
    
		ACC[2] += RC * 0.13015383f;

		ACC[10] += RC * -0.27304852f;

		ACC[15] += RC * 0.12982921f;

		ACC[17] += RC * -0.09169744f;

		ACC[22] += RC * -0.11175012f;

		ACC[29] += RC * -0.17992558f;

		ACC[38] += RC * -0.10121323f;

		ACC[39] += RC * 0.05587235f;

		ACC[41] += RC * 0.25168929f;

		ACC[46] += RC * -0.1194533f;

		    RC = BC[68992 + C_offset + lane];
    
		ACC[2] += RC * 0.037355147f;

		ACC[29] += RC * 0.24902283f;

		    RC = BC[72128 + C_offset + lane];
    
		ACC[15] += RC * -0.28222182f;

		ACC[46] += RC * 0.03862857f;

		    RC = BC[75264 + C_offset + lane];
    
		ACC[17] += RC * 0.13590872f;

		ACC[23] += RC * 0.9363806f;

		ACC[25] += RC * 0.09702684f;

		ACC[28] += RC * 0.066035986f;

		ACC[44] += RC * 0.18337922f;

		    RC = BC[78400 + C_offset + lane];
    
		ACC[17] += RC * 0.08865731f;

		ACC[49] += RC * -0.6772053f;

		    RC = BC[81536 + C_offset + lane];
    
		ACC[1] += RC * 0.085653275f;

		ACC[4] += RC * 0.44263017f;

		ACC[20] += RC * -0.21628965f;

		ACC[32] += RC * -0.38544872f;

		ACC[34] += RC * 0.37335655f;

		ACC[40] += RC * -0.3120128f;

		    RC = BC[84672 + C_offset + lane];
    
		ACC[0] += RC * -0.20678516f;

		ACC[3] += RC * 0.32530537f;

		ACC[7] += RC * -0.64888483f;

		ACC[16] += RC * 0.24505733f;

		ACC[18] += RC * 0.5256636f;

		ACC[19] += RC * -0.3272166f;

		ACC[22] += RC * -0.13409023f;

		ACC[26] += RC * -0.17373987f;

		ACC[31] += RC * 0.2050952f;

		ACC[47] += RC * 0.11039164f;

		ACC[48] += RC * -0.2266204f;

		ACC[54] += RC * -0.21862362f;

		ACC[59] += RC * 0.12699634f;

		    RC = BC[87808 + C_offset + lane];
    
		ACC[9] += RC * 0.15279768f;

		ACC[11] += RC * 0.16959256f;

		ACC[16] += RC * -0.42881632f;

		ACC[59] += RC * 0.14944461f;

		    RC = BC[90944 + C_offset + lane];
    
		ACC[3] += RC * 0.33154777f;

		ACC[9] += RC * -0.17489237f;

		ACC[34] += RC * -0.23416986f;

		ACC[38] += RC * -0.11405346f;

		ACC[41] += RC * -0.121796f;

		ACC[47] += RC * 0.16470617f;

		ACC[52] += RC * 0.21700798f;

		ACC[54] += RC * 0.15729876f;

		ACC[56] += RC * 0.23263544f;

		ACC[59] += RC * -0.21623065f;

		ACC[61] += RC * 0.60681957f;

		ACC[62] += RC * -0.2807839f;

		ACC[63] += RC * 0.13468657f;

		    RC = BC[94080 + C_offset + lane];
    
		ACC[15] += RC * -0.28389496f;

		ACC[22] += RC * -0.23826738f;

		ACC[46] += RC * 0.08869904f;

		    RC = BC[97216 + C_offset + lane];
    
		ACC[15] += RC * -0.101051666f;

		ACC[21] += RC * -0.20413171f;

		ACC[46] += RC * 0.0930694f;

		    RC = BC[100352 + C_offset + lane];
    
		ACC[49] += RC * -0.46746957f;

		    RC = BC[103488 + C_offset + lane];
    
		ACC[7] += RC * -0.23927209f;

		ACC[14] += RC * 0.39993063f;

		ACC[28] += RC * 0.06608304f;

		ACC[31] += RC * 0.41690415f;

		ACC[41] += RC * -0.16268021f;

		ACC[47] += RC * -0.06665658f;

		ACC[54] += RC * 0.3898219f;

		ACC[58] += RC * -0.4811902f;

		ACC[62] += RC * -0.20769393f;

		    RC = BC[106624 + C_offset + lane];
    
		ACC[4] += RC * 0.23032154f;

		ACC[9] += RC * 0.23267138f;

		ACC[18] += RC * 0.17405392f;

		ACC[19] += RC * -0.30409917f;

		ACC[26] += RC * -0.3498056f;

		ACC[33] += RC * -0.41797748f;

		ACC[37] += RC * -0.14948401f;

		ACC[40] += RC * -0.7528111f;

		ACC[47] += RC * -0.12718853f;

		ACC[59] += RC * 0.30002242f;

		ACC[60] += RC * 0.23654762f;

		    RC = BC[109760 + C_offset + lane];
    
		ACC[2] += RC * 0.029297013f;

		ACC[26] += RC * 0.14118852f;

		ACC[28] += RC * -0.09510913f;

		ACC[33] += RC * 0.24073741f;

		ACC[46] += RC * -0.07880714f;

		    RC = BC[112896 + C_offset + lane];
    
		ACC[13] += RC * 0.47092745f;

		ACC[25] += RC * 0.14386271f;

		ACC[35] += RC * 0.6366757f;

		ACC[44] += RC * -0.27373043f;

		ACC[50] += RC * -0.7100229f;

		ACC[57] += RC * 0.60796255f;

		    RC = BC[116032 + C_offset + lane];
    
		ACC[8] += RC * 0.25282413f;

		ACC[18] += RC * -0.106563374f;

		ACC[28] += RC * -0.16593163f;

		ACC[41] += RC * 0.30251354f;

		ACC[47] += RC * 0.13658065f;

		ACC[54] += RC * -0.69706804f;

		    RC = BC[119168 + C_offset + lane];
    
		ACC[19] += RC * -0.1432965f;

		ACC[31] += RC * 0.29604664f;

		ACC[34] += RC * -0.25980312f;

		ACC[38] += RC * 0.12999922f;

		ACC[47] += RC * -0.0854889f;

		ACC[49] += RC * -0.29304054f;

		ACC[54] += RC * -0.27289852f;

		ACC[56] += RC * 0.20387344f;

		ACC[58] += RC * 0.23806095f;

		ACC[60] += RC * -0.18487847f;

		ACC[63] += RC * 0.38124996f;

		    RC = BC[122304 + C_offset + lane];
    
		ACC[22] += RC * -0.32027736f;

		    RC = BC[125440 + C_offset + lane];
    
		ACC[1] += RC * -0.1730987f;

		ACC[2] += RC * 0.06058641f;

		ACC[11] += RC * -0.47734895f;

		ACC[20] += RC * -0.3253965f;

		ACC[27] += RC * 0.23532234f;

		ACC[29] += RC * 0.28981793f;

		ACC[32] += RC * 0.27740094f;

		ACC[38] += RC * 0.08028157f;

		ACC[41] += RC * 0.2635885f;

		ACC[43] += RC * -0.30340588f;

		ACC[46] += RC * -0.06796846f;

		ACC[51] += RC * -0.3080269f;

		ACC[52] += RC * -0.3164784f;

		ACC[54] += RC * -0.1577772f;

		    RC = BC[128576 + C_offset + lane];
    
		ACC[1] += RC * 0.17775148f;

		ACC[2] += RC * -0.052164525f;

		ACC[17] += RC * -0.14576583f;

		ACC[29] += RC * -0.30311817f;

		ACC[38] += RC * -0.10872306f;

		ACC[39] += RC * 0.08131772f;

		ACC[41] += RC * -0.18844928f;

		ACC[51] += RC * 0.16056864f;

		    RC = BC[131712 + C_offset + lane];
    
		ACC[6] += RC * 0.13167883f;

		ACC[17] += RC * 0.098335385f;

		ACC[31] += RC * -0.37775412f;

		ACC[38] += RC * 0.1992557f;

		ACC[39] += RC * -0.029296694f;

		ACC[41] += RC * 0.142547f;

		ACC[47] += RC * -0.08436086f;

		ACC[49] += RC * -0.31818718f;

		ACC[54] += RC * -0.4348505f;

		ACC[55] += RC * 0.5758083f;

		ACC[58] += RC * -0.31610832f;

		ACC[62] += RC * -0.28211477f;

		ACC[63] += RC * -0.088721775f;

		    RC = BC[134848 + C_offset + lane];
    
		ACC[1] += RC * 0.16146538f;

		ACC[2] += RC * -0.0475772f;

		ACC[6] += RC * 0.3263237f;

		ACC[8] += RC * -0.6276081f;

		ACC[9] += RC * 0.11742934f;

		ACC[10] += RC * 0.20307045f;

		ACC[17] += RC * 0.12657759f;

		ACC[28] += RC * 0.10099993f;

		ACC[29] += RC * -0.24853194f;

		ACC[38] += RC * 0.13664165f;

		ACC[39] += RC * -0.059467845f;

		ACC[41] += RC * 0.27962637f;

		ACC[47] += RC * -0.33069918f;

		ACC[62] += RC * 0.4070311f;

		    RC = BC[137984 + C_offset + lane];
    
		ACC[5] += RC * -0.5277919f;

		ACC[13] += RC * 0.4151213f;

		ACC[23] += RC * 0.3098378f;

		ACC[50] += RC * 0.60081923f;

		ACC[53] += RC * 0.9438657f;

		ACC[57] += RC * -0.48226568f;

		ACC[62] += RC * -0.12101145f;

		    RC = BC[141120 + C_offset + lane];
    
		ACC[1] += RC * -0.22886378f;

		ACC[2] += RC * 0.12361719f;

		ACC[4] += RC * -0.72761476f;

		ACC[20] += RC * 0.07841185f;

		ACC[27] += RC * 0.1727555f;

		ACC[29] += RC * 0.22075741f;

		ACC[32] += RC * 0.24208428f;

		ACC[38] += RC * 0.18292496f;

		ACC[41] += RC * 0.30513752f;

		ACC[45] += RC * -0.2635817f;

		ACC[46] += RC * -0.0830069f;

		ACC[52] += RC * -0.44706273f;

		ACC[54] += RC * -0.17014933f;

		    RC = BC[144256 + C_offset + lane];
    
		ACC[12] += RC * 0.2964925f;

		ACC[29] += RC * 0.26264462f;

		ACC[32] += RC * 0.14424336f;

		    RC = BC[147392 + C_offset + lane];
    
		ACC[16] += RC * 0.27318072f;

		ACC[18] += RC * -0.17249444f;

		ACC[19] += RC * 0.1799758f;

		ACC[42] += RC * 0.37337103f;

		ACC[54] += RC * -0.13773881f;

		ACC[60] += RC * -0.15917754f;

		    RC = BC[150528 + C_offset + lane];
    
		ACC[24] += RC * 0.8332341f;

		ACC[25] += RC * -0.12432293f;

		ACC[30] += RC * -1.3509405f;

		ACC[44] += RC * 0.37730205f;

		    RC = BC[153664 + C_offset + lane];
    
		ACC[17] += RC * -0.080590144f;

		ACC[28] += RC * -0.067727126f;

		ACC[38] += RC * -0.11767121f;

		ACC[47] += RC * -0.053688377f;

		    RC = BC[156800 + C_offset + lane];
    
		ACC[6] += RC * 0.17148225f;

		ACC[19] += RC * 0.15576942f;

		ACC[34] += RC * 0.24993496f;

		ACC[43] += RC * 0.28970215f;

		ACC[47] += RC * -0.049092744f;

		ACC[48] += RC * -0.2749422f;

		ACC[49] += RC * -0.482139f;

		ACC[54] += RC * -0.21819173f;

		ACC[56] += RC * -0.2958262f;

		ACC[60] += RC * 0.1939095f;

		    RC = BC[159936 + C_offset + lane];
    
		ACC[11] += RC * -0.20927271f;

		ACC[16] += RC * -0.47554716f;

		    RC = BC[163072 + C_offset + lane];
    
		ACC[25] += RC * -0.04016468f;

		ACC[26] += RC * 0.23068126f;

		ACC[33] += RC * 0.4181578f;

		ACC[36] += RC * -1.1136609f;

		ACC[37] += RC * 0.097754344f;

		ACC[60] += RC * -0.32576373f;

		    RC = BC[166208 + C_offset + lane];
    
		ACC[13] += RC * -0.30980462f;

		ACC[23] += RC * -0.4268186f;

		ACC[50] += RC * -0.28890938f;

		ACC[57] += RC * 0.23223265f;

		    RC = BC[172480 + C_offset + lane];
    
		ACC[1] += RC * -0.16658776f;

		ACC[2] += RC * 0.0996416f;

		ACC[9] += RC * 0.19291757f;

		ACC[11] += RC * 0.63438296f;

		ACC[20] += RC * -0.1459365f;

		ACC[27] += RC * 0.11957474f;

		ACC[29] += RC * -0.24668978f;

		ACC[46] += RC * -0.08030703f;

		ACC[52] += RC * -0.3264767f;

		ACC[60] += RC * -0.21778284f;

		    RC = BC[175616 + C_offset + lane];
    
		ACC[17] += RC * 0.19204687f;

		ACC[38] += RC * 0.1554395f;

		ACC[39] += RC * -0.047485f;

		ACC[41] += RC * 0.11880568f;

		ACC[54] += RC * -0.23030247f;

		    RC = BC[178752 + C_offset + lane];
    
		ACC[12] += RC * -0.66184276f;

		ACC[32] += RC * -0.33932766f;

		ACC[42] += RC * -0.2929955f;

		ACC[51] += RC * 0.20616594f;

		ACC[52] += RC * 0.1678827f;

		    RC = BC[181888 + C_offset + lane];
    
		ACC[2] += RC * 0.043896176f;

		ACC[22] += RC * -0.20326512f;

		ACC[26] += RC * -0.21282434f;

		ACC[28] += RC * -0.08408172f;

		ACC[33] += RC * -0.3274323f;

		ACC[54] += RC * -0.23184451f;

		ACC[60] += RC * 0.25018582f;

		    RC = BC[185024 + C_offset + lane];
    
		ACC[15] += RC * -0.23927557f;

		ACC[22] += RC * -0.112599134f;

		ACC[46] += RC * 0.10714278f;

		    RC = BC[188160 + C_offset + lane];
    
		ACC[2] += RC * -0.06315396f;

		ACC[9] += RC * 0.3384536f;

		ACC[22] += RC * 0.15122199f;

		ACC[29] += RC * -0.27005216f;

		ACC[31] += RC * -0.11214516f;

		ACC[41] += RC * 0.27858534f;

		ACC[52] += RC * -0.15154484f;

		ACC[54] += RC * -0.2562287f;

		ACC[63] += RC * -0.10051341f;

		    RC = BC[191296 + C_offset + lane];
    
		ACC[2] += RC * -0.07809353f;

		ACC[6] += RC * 0.12629469f;

		ACC[10] += RC * 0.25280967f;

		ACC[15] += RC * -0.36020976f;

		ACC[21] += RC * 0.6824156f;

		ACC[29] += RC * -0.18479209f;

		ACC[41] += RC * -0.10888942f;

		ACC[46] += RC * 0.12745552f;

		ACC[52] += RC * 0.16152337f;

		ACC[54] += RC * 0.13135277f;

		    RC = BC[194432 + C_offset + lane];
    
		ACC[2] += RC * 0.020466007f;

		ACC[6] += RC * -0.27377614f;

		ACC[7] += RC * 0.1725072f;

		ACC[21] += RC * 0.7983611f;

		ACC[28] += RC * -0.088834435f;

		ACC[29] += RC * 0.3583734f;

		ACC[38] += RC * -0.08311054f;

		ACC[52] += RC * -0.3431529f;

		ACC[54] += RC * -0.22426634f;

		    RC = BC[197568 + C_offset + lane];
    
		ACC[1] += RC * 0.13151796f;

		ACC[17] += RC * -0.07872371f;

		ACC[28] += RC * -0.08544529f;

		ACC[39] += RC * 0.08887849f;

	}


	if(groupId == 1)
	{

		for (int b_idx = 0; b_idx < 64; b_idx++)
		{
		for (int ny_idx = block_NY/2; ny_idx < block_NY; ny_idx++)
		{
		int a_idx = 0 + ny_idx;
		RC = BC[b_idx * 3136 + lane];
		ACC[ny_idx] += RC * BA[b_idx, a_idx];
		}
		}

	}



        AC[0 + C_offset  + lane] = max(ACC[0] + 3.977744f,0.0f);

        AC[3136 + C_offset  + lane] = max(ACC[1] + 6.530452f,0.0f);

        AC[6272 + C_offset  + lane] = max(ACC[2] + 0.6891806f,0.0f);

        AC[9408 + C_offset  + lane] = max(ACC[3] + 1.9361705f,0.0f);

        AC[12544 + C_offset  + lane] = max(ACC[4] + -3.6765223f,0.0f);

        AC[15680 + C_offset  + lane] = max(ACC[5] + 5.3994093f,0.0f);

        AC[18816 + C_offset  + lane] = max(ACC[6] + 2.343494f,0.0f);

        AC[21952 + C_offset  + lane] = max(ACC[7] + 4.982128f,0.0f);

        AC[25088 + C_offset  + lane] = max(ACC[8] + -0.9497072f,0.0f);

        AC[28224 + C_offset  + lane] = max(ACC[9] + -0.8413665f,0.0f);

        AC[31360 + C_offset  + lane] = max(ACC[10] + 10.717709f,0.0f);

        AC[34496 + C_offset  + lane] = max(ACC[11] + -0.6981136f,0.0f);

        AC[37632 + C_offset  + lane] = max(ACC[12] + 2.9473062f,0.0f);

        AC[40768 + C_offset  + lane] = max(ACC[13] + -1.6245604f,0.0f);

        AC[43904 + C_offset  + lane] = max(ACC[14] + 0.061071157f,0.0f);

        AC[47040 + C_offset  + lane] = max(ACC[15] + -2.6851745f,0.0f);

        AC[50176 + C_offset  + lane] = max(ACC[16] + 2.9532514f,0.0f);

        AC[53312 + C_offset  + lane] = max(ACC[17] + 2.908723f,0.0f);

        AC[56448 + C_offset  + lane] = max(ACC[18] + 0.58572805f,0.0f);

        AC[59584 + C_offset  + lane] = max(ACC[19] + 2.4610727f,0.0f);

        AC[62720 + C_offset  + lane] = max(ACC[20] + 5.1531906f,0.0f);

        AC[65856 + C_offset  + lane] = max(ACC[21] + -21.59833f,0.0f);

        AC[68992 + C_offset  + lane] = max(ACC[22] + 6.769128f,0.0f);

        AC[72128 + C_offset  + lane] = max(ACC[23] + -1.4368893f,0.0f);

        AC[75264 + C_offset  + lane] = max(ACC[24] + -2.4415185f,0.0f);

        AC[78400 + C_offset  + lane] = max(ACC[25] + 2.3887634f,0.0f);

        AC[81536 + C_offset  + lane] = max(ACC[26] + 4.2395906f,0.0f);

        AC[84672 + C_offset  + lane] = max(ACC[27] + 1.7414098f,0.0f);

        AC[87808 + C_offset  + lane] = max(ACC[28] + 4.020364f,0.0f);

        AC[90944 + C_offset  + lane] = max(ACC[29] + 3.3077111f,0.0f);

        AC[94080 + C_offset  + lane] = max(ACC[30] + 1.8318611f,0.0f);

        AC[97216 + C_offset  + lane] = max(ACC[31] + -3.4719281f,0.0f);

        AC[100352 + C_offset  + lane] = max(ACC[32] + 1.4029231f,0.0f);

        AC[103488 + C_offset  + lane] = max(ACC[33] + 2.0068154f,0.0f);

        AC[106624 + C_offset  + lane] = max(ACC[34] + 0.9218654f,0.0f);

        AC[109760 + C_offset  + lane] = max(ACC[35] + -6.0270934f,0.0f);

        AC[112896 + C_offset  + lane] = max(ACC[36] + 3.3614318f,0.0f);

        AC[116032 + C_offset  + lane] = max(ACC[37] + 2.8095539f,0.0f);

        AC[119168 + C_offset  + lane] = max(ACC[38] + 3.403461f,0.0f);

        AC[122304 + C_offset  + lane] = max(ACC[39] + 8.1054325f,0.0f);

        AC[125440 + C_offset  + lane] = max(ACC[40] + 3.8145614f,0.0f);

        AC[128576 + C_offset  + lane] = max(ACC[41] + -7.17782f,0.0f);

        AC[131712 + C_offset  + lane] = max(ACC[42] + 1.0847245f,0.0f);

        AC[134848 + C_offset  + lane] = max(ACC[43] + 1.5190965f,0.0f);

        AC[137984 + C_offset  + lane] = max(ACC[44] + -1.1734383f,0.0f);

        AC[141120 + C_offset  + lane] = max(ACC[45] + 0.07577264f,0.0f);

        AC[144256 + C_offset  + lane] = max(ACC[46] + 7.681935f,0.0f);

        AC[147392 + C_offset  + lane] = max(ACC[47] + 4.0016623f,0.0f);

        AC[150528 + C_offset  + lane] = max(ACC[48] + 4.7618046f,0.0f);

        AC[153664 + C_offset  + lane] = max(ACC[49] + 8.247117f,0.0f);

        AC[156800 + C_offset  + lane] = max(ACC[50] + 10.89353f,0.0f);

        AC[159936 + C_offset  + lane] = max(ACC[51] + 1.5308125f,0.0f);

        AC[163072 + C_offset  + lane] = max(ACC[52] + 5.9652867f,0.0f);

        AC[166208 + C_offset  + lane] = max(ACC[53] + 2.222279f,0.0f);

        AC[169344 + C_offset  + lane] = max(ACC[54] + 16.071999f,0.0f);

        AC[172480 + C_offset  + lane] = max(ACC[55] + 4.40592f,0.0f);

        AC[175616 + C_offset  + lane] = max(ACC[56] + 3.53694f,0.0f);

        AC[178752 + C_offset  + lane] = max(ACC[57] + -6.28261f,0.0f);

        AC[181888 + C_offset  + lane] = max(ACC[58] + 4.730913f,0.0f);

        AC[185024 + C_offset  + lane] = max(ACC[59] + 0.80060506f,0.0f);

        AC[188160 + C_offset  + lane] = max(ACC[60] + 4.3881507f,0.0f);

        AC[191296 + C_offset  + lane] = max(ACC[61] + 2.3128614f,0.0f);

        AC[194432 + C_offset  + lane] = max(ACC[62] + 4.072897f,0.0f);

        AC[197568 + C_offset  + lane] = max(ACC[63] + 4.2837954f,0.0f);
for (int i = block_NY/2; i < block_NY; i++)
{

        AC[(A_offset + i) * 3136 + C_offset  + lane] = max(ACC[i] + bias[A_offset+i],0.0f);
}
}

}
int main()
{

	std::cout << "Group size " << Gsy << std::endl;

	cnpy::NpyArray arr = cnpy::npy_load("mobilenet/contraction_1x1_1_transposed.npy");
	float * AB = arr.data<float>();
	assert(arr.word_size = sizeof(float));
	assert(arr.shape.size()==2 && arr.shape[0] == 64 && arr.shape[1] == 128); //transposed

	cnpy::NpyArray arr1 = cnpy::npy_load("BC.npy");
	float * BC = arr1.data<float>();
	assert(arr1.word_size = sizeof(float));
#if In_Format == 'NHWC'
	assert(arr1.shape.size()==2 && arr1.shape[0] == 3136 && arr1.shape[1] == 64);
#else
	assert(arr1.shape.size()==2 && arr1.shape[0] == 64 && arr1.shape[1] == 3136);
#endif
	
	cnpy::NpyArray arr4 = cnpy::npy_load("mobilenet/contraction_1x1_1_bias.npy");
	float * bias = arr4.data<float>();
	assert(arr4.word_size = sizeof(float));
#if In_Format == 'NHWC'
	assert(arr4.shape.size()==1 && arr4.shape[0] == 128);
#else
	assert(arr4.shape.size()==1 && arr4.shape[0] == 128);
#endif
	
    cnpy::NpyArray arr2 = cnpy::npy_load("ref.npy");
	float * AC = arr2.data<float>();
    std::cout << AC[0] << std::endl;

	float *d_BC, *d_AC;
	float *d_BA, *d_bias;
	hipMalloc((void**)&d_BC, 64 * 3136 *sizeof(float));
	hipMalloc((void**)&d_AC, 128 * 3136 *sizeof(float));
	hipMalloc((void**)&d_BA, 64 * 128 *sizeof(float));
	hipMalloc((void**)&d_bias, 64 * 3136 *sizeof(float));


	hipMemcpy( d_BC,BC, 64 * 3136 *sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy( d_bias,bias, 64 * 3136 *sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy( d_BA,AB, 64 * 128 *sizeof(float), hipMemcpyHostToDevice);

	float *result;
	result = (float *)malloc(128 * 3136 *sizeof(result));

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	dim3 GS(1,98);

     std::cout << "warning: sometimes you might want to fix the launch dimensions to 32" << std::endl;
    // We now launch twice the number of threads
    for(int i = 0;i < 1000;i ++){
	    mm<<<GS,Gsy * 2>>>(d_BC,d_AC,d_BA,d_bias);
    }

	hipProfilerStart();
	hipEventRecord(start);

	for(int i = 0;i < 1000;i ++){
	    mm<<<GS,Gsy * 2>>>(d_BC,d_AC,d_BA,d_bias);
    }
	hipEventRecord(stop);
	hipProfilerStop();
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	std::cout << "kernel used " << time / 1000.0 << std::endl;


	hipMemcpy(result, d_AC, 128 * 3136 *sizeof(float), hipMemcpyDeviceToHost);

	float error = 0;
	for(int i = 0 ; i < 128 * 3136; i ++)
	{
        error += abs(result[i] - AC[i]);
	}
	
	#if Out_Format == 'NCHW'
        cnpy::npy_save("result.npy",&result[0],{128,3136},"w");
    #else
        cnpy::npy_save("result.npy",&result[0],{3136,128},"w");
    #endif

	std::cout << result[0] << result[1] << result[2] << std::endl;
	std::cout << error << std::endl;
	hipFree(d_BC);
	hipFree(d_AC);
}
